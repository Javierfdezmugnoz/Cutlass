
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

typedef float    float32_t;
typedef double   float64_t;
typedef void     void_t;
typedef __uint32_t uint32_t;

__device__ uint32_t __xor (uint32_t ui32_a, uint32_t ui32_b)
{
    uint32_t acc;
    asm ("xor.b32         %0, %1, %2;\n\t"
                : "=r"(acc)
                : "r"(ui32_a), "r"(ui32_b));
    return acc;
}

/* ==========================================================================
  Descritption: Addition of two values using PTX (parallel thread execution) 
  and ISA (parallel thread execution with instruction set architecture) adding
  the carry bit. After that, one's complement is implemented (bit negation). 
  Additional info:
  url: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html or
  https://docs.nvidia.com/pdf/ptx_isa_5.0.pdf 
=============================================================================*/
__device__ uint32_t __a1c (uint32_t ui32_a, uint32_t ui32_b)
{
    uint32_t acc;
    asm ("add.cc.u32      %0, %1, %2;\n\t"
         "addc.u32        %0, %0, 0;\n\t"
         "not.b32         %0, %0;\n\t"
         : "=r"(acc)
         : "r"(ui32_a), "r"(ui32_b));
    return acc;
}

__device__ uint32_t __a2c (uint32_t ui32_a, uint32_t ui32_b)
{
    uint32_t acc = 0;
    asm volatile ("add.u32     %0, %1, %2;\n\t"
         "not.b32     %0, %0;\n\t"
         "add.u32     %0, %0, 1;\n\t"
         : "=r"(acc)
         : "r"(ui32_a), "r"(ui32_b));
    return acc;
}


__global__ void kernel_ES (uint32_t *a_val, uint32_t *b_val,uint32_t *ES_val){
    ES_val[0] = __xor(a_val[0],b_val[0]);
    printf("\ta_val: %x \t b_val: %x\t ES_xor = %x\n", (uint32_t) *((uint32_t*) &a_val[0]),(uint32_t) *((uint32_t*) &b_val[0]),(uint32_t) *((uint32_t*) &ES_val[0]));
    ES_val[0] = __a2c(a_val[0],b_val[0]);
    printf("\ta_val: %x \t b_val: %x\t ES_a2  = %x\n", (uint32_t) *((uint32_t*) &a_val[0]),(uint32_t) *((uint32_t*) &b_val[0]),(uint32_t) *((uint32_t*) &ES_val[0]));
    ES_val[0] = __a1c(a_val[0],b_val[0]);
    printf("\ta_val: %x \t b_val: %x\t ES_a1  = %x\n", (uint32_t) *((uint32_t*) &a_val[0]),(uint32_t) *((uint32_t*) &b_val[0]),(uint32_t) *((uint32_t*) &ES_val[0]));

    //*ES_val = __xor((uint32_t) *((uint32_t*) &a_val[0]),(uint32_t) *((uint32_t*) &b_val[0]));
    // *ES_val = __a1c(u32_a_value,u32_b_value);
    return;
}

int main(void){
    // Define pointers to ES_a, ES_b
    uint32_t *h_ES_a;
    uint32_t *h_ES_b;
    uint32_t *h_ES_ES;

    // Allocate h_ES_a and h_ES_b
    h_ES_a = (uint32_t *) malloc(sizeof(uint32_t));
    h_ES_b = (uint32_t *) malloc(sizeof(uint32_t));
    h_ES_ES= (uint32_t *) malloc(sizeof(uint32_t));

    // Initialize values of h_ES_a and h_ES_b
    *h_ES_a = 0xfffeffee;
    *h_ES_b = 0x00101111;
    *h_ES_ES = 0u;    

    // Define pointers to d_ES_a,b
    uint32_t *d_ES_a;
    uint32_t *d_ES_b;
    uint32_t *d_ES_ES;

    // Allocate d_ES_a,b,c in GPU
    hipError_t result;
    result = hipMalloc((uint32_t **) &d_ES_a, sizeof(uint32_t));
    if (result != hipSuccess) {
        std::cerr << "Failed to Allocate d_ES_a: "<< hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        return result;
    }
    result = hipMalloc((uint32_t **) &d_ES_b, sizeof(uint32_t));
    if (result != hipSuccess) {
        std::cerr << "Failed to Allocate d_ES_b: "<< hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        return result;
    }

    result = hipMalloc((uint32_t **) &d_ES_ES, sizeof(uint32_t));
    if (result != hipSuccess) {
        std::cerr << "Failed to Allocate d_ES_b: "<< hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        return result;
    }

    // Initialize d_ES_a and d_ES_b
    result = hipMemcpy(d_ES_a, h_ES_a, sizeof(uint32_t), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_ES_a matrix to d_ES_a: " << hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        return result;
    }

    result = hipMemcpy(d_ES_b, h_ES_b, sizeof(uint32_t), hipMemcpyHostToDevice);
    if (result != hipSuccess) 
    {
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        return result;
    }


    printf("Here we go!\n");
    dim3 block(1,1,1);
    dim3 grid(1,1,1);
    printf("u32_a_value: %x \tu32_b_value[1]: %x\t Xor = %x\n", *h_ES_b,*h_ES_a,*h_ES_ES);

    kernel_ES<<< grid, block >>>(d_ES_a,d_ES_b,d_ES_ES);
    result= hipGetLastError();
    if (result != hipSuccess) {
      std::cerr << "Failed execute kernel_ES" << hipGetErrorString(result) << std::endl;
      hipFree(h_ES_a);
      hipFree(h_ES_b);
      return result;
    }


    // Copy to host the values of the ES of A, B and C performed and stored in the GPU device
    result = hipMemcpy(h_ES_a, d_ES_a, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_a_DC_a matrix to h_ES_a (return)" << hipGetErrorString(result) << std::endl;
      hipFree(h_ES_a);
      hipFree(h_ES_b);
      return result;
    }

    result = hipMemcpy(h_ES_b, d_ES_b, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_b_DC_a matrix to h_ES_b (return)" << hipGetErrorString(result) << std::endl;
      hipFree(h_ES_a);
      hipFree(h_ES_b);
      return result;
    }

    result = hipMemcpy(h_ES_ES, d_ES_ES, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_b_DC_a matrix to h_ES_b (return)" << hipGetErrorString(result) << std::endl;
      hipFree(h_ES_a);
      hipFree(h_ES_b);
      hipFree(h_ES_ES);
      return result;
    }
    printf("u32_a_value: %x \tu32_b_value[1]: %x\t Xor = %x\n", h_ES_b[0],h_ES_a[0],h_ES_ES[0]);
    return 0;
}