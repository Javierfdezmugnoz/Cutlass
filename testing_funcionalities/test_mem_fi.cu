
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef float    float32_t;
typedef double   float64_t;
typedef void     void_t;
typedef __uint32_t uint32_t;

static void_t mem_fi(float32_t* const paf32_m, uint32_t ui32_bit_idx)
{
	uint32_t ui32_idx_flt = ui32_bit_idx / (sizeof(float32_t)*CHAR_BIT),
		ui32_idx_flt_bit = ui32_bit_idx % (sizeof(float32_t)*CHAR_BIT);
	uint32_t ui32_f_d = *((uint32_t *)&paf32_m[ui32_idx_flt]);

	ui32_f_d ^= (1u << ui32_idx_flt_bit);
	paf32_m[ui32_idx_flt] = *((float32_t *)&ui32_f_d);
}

int main(void){
    float32_t f32_var1[3] = {0};
    for (__uint32_t ui32_loop_i=0; ui32_loop_i<64; ui32_loop_i++)
    {
        printf("value of ui32_var1[0]: %x ui32_var1[1]: %x ui32_var1[2]: %x\n",(uint32_t) *((uint32_t*) &f32_var1[0]), (uint32_t) *((uint32_t*) &f32_var1[1]),(uint32_t) *((uint32_t*) &f32_var1[2]));
        mem_fi(&f32_var1[0],ui32_loop_i);
        printf("value of ui32_var1[0]: %x ui32_var1[1]: %x ui32_var1[2]: %x\n\n",(uint32_t) *((uint32_t*) &f32_var1[0]),(uint32_t) *((uint32_t*) &f32_var1[1]),(uint32_t) *((uint32_t*) &f32_var1[2]));
        mem_fi(&f32_var1[0],ui32_loop_i);
    }
    return 0;
}