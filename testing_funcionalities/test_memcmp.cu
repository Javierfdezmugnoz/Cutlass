
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

int  main(){
	// TEST 01: memset behaviour
	__uint32_t nElem_ES = 10;	
	__uint32_t *h_value_1;
	__uint32_t *h_value_2;
	__uint32_t *h_value_3;
	__uint32_t nBytes = nElem_ES * sizeof(__uint32_t);

	h_value_1 = (__uint32_t *) malloc(nBytes);
	h_value_2 = (__uint32_t *) malloc(nBytes);
	h_value_3 = (__uint32_t *) malloc(nBytes);
	memset(h_value_1, 0 , nBytes);
	memset(h_value_2, 0 , nBytes);
	memset(h_value_3, 0 , nBytes);

	printf("TEST 01: Verify behaviour of memcmp:\n");
	if(memcmp(h_value_1,h_value_2,nBytes)!=0){
		printf("Memcmp detects 1)\n");
	}

	if((memcmp(h_value_1,h_value_2,nBytes)!=0) || (memcmp(h_value_1,h_value_3,nBytes)!=0)){
		printf("Memcmp detects the change 4)\n");
	}


	h_value_1[0]=1;
	if(memcmp(h_value_1,h_value_2,nBytes)!=0){
		printf("Memcmp detects 2)\n");
	}

	if((memcmp(h_value_1,h_value_2,nBytes)!=0) || (memcmp(h_value_1,h_value_3,nBytes)!=0)){
		printf("Memcmp detects the change 3)\n");
	}



	printf("TEST 01\n");
	/*
	if(*h_value_1 != *h_value_2){
		printf("After memset");
	}
	if(memcmp(h_value_1,h_value_2,nBytes)!=0){
		printf("Memcmp detects the change");
	}
	
	h_value_1[1]= 11u;
	if(*h_value_1 != *h_value_2){
		printf("I have modified the value of h_1[1]");
	}
	if(memcmp(h_value_1,h_value_2,nBytes)!=0){
		printf("Memcmp detects the change o h_1[1]");
	}

	h_value_1[0]= 13u;
	if(*h_value_1 != *h_value_2){
		printf("I have modified the value of h_1[0]");
	}
	if(memcmp(h_value_1,h_value_2,nBytes)!=0){
		printf("Memcmp detects the change of h_1[0]");
	}

	h_value_2[0]= 13u;
	h_value_2[1]= 11u;
	if(*h_value_1 != *h_value_2){
		printf("I have modified the values of h_2");
	}
	if(memcmp(h_value_1,h_value_2,nBytes)!=0){
		printf("Memcmp no detects");
	}
	*/
	
	// Test 02: 
	__uint32_t mod_operand = 123541;
	__uint32_t mod_divisor = 256;
	__uint32_t out_mod = 0;
	__uint32_t out_var = 0;
	out_mod = mod_operand % mod_divisor;
	out_var = (mod_operand & (mod_divisor-1));
	printf("value of mod_operand: %u\nvalue of var: %u\n",out_mod,out_var);

return 0;

}
