#include "hip/hip_runtime.h"
#pragma once
/* ==============================================================================================================
* 												INCLUDES
* ============================================================================================================== */
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <stdint.h>
#include "checksum.h"

/* ==========================================================================
  Descritption: Addition of two values using PTX (parallel thread execution) 
  and ISA (parallel thread execution with instruction set architecture) adding
  the carry bit. After that, one's complement is implemented (bit negation). 
  Additional info:
  url: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html or
  https://docs.nvidia.com/pdf/ptx_isa_5.0.pdf 
=============================================================================*/
__device__ uint32_t __a1c (uint32_t ui32_a, uint32_t ui32_b)
{
    uint32_t acc;
    asm ("add.cc.u32      %0, %1, %2;\n\t"
         "addc.u32        %0, %0, 0;\n\t"
         "not.b32         %0, %0;\n\t"
         : "=r"(acc)
         : "r"(ui32_a), "r"(ui32_b));
    return acc;
}


/* ==========================================================================
  Descritption: Ones complement with atomic instructions
=============================================================================*/
__device__ uint32_t a1c_atomic (uint32_t ui32_a, uint32_t ui32_b)
{
    atomicAdd((uint32_t*) &ui32_a, ui32_b);
    ui32_a = ~ui32_a;
    return ui32_a;
}


/* ==========================================================================
  Description: Addition of two values using PTX (parallel thread execution) 
  and ISA (parallel thread execution with instruction set architecture) with
  not carry-bit addition. This operation is followed by the two's complement
  implementation (bit negation and then, addition of 1)
  Additional info:
  url: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html or
  https://docs.nvidia.com/pdf/ptx_isa_5.0.pdf
=============================================================================*/
__device__ uint32_t __a2c (uint32_t ui32_a, uint32_t ui32_b)
{
    uint32_t acc = 0;
    asm ("add.u32     %0, %1, %2;\n\t"
         "not.b32     %0, %0;\n\t"
         "add.u32     %0, %0, 1;\n\t"
         : "=r"(acc)
         : "r"(ui32_a), "r"(ui32_b));
    return acc;
}

/* ==========================================================================
  Descritption: Two's complement with atomic instructions
=============================================================================*/
__device__ uint32_t a2c_atomic (uint32_t ui32_a, uint32_t ui32_b)
{
    atomicAdd((uint32_t*) &ui32_a, ui32_b);
    ui32_a = ~ui32_a;
    atomicAdd((uint32_t*) &ui32_a, 1u);
    return ui32_a;
}

/* ==========================================================================
  Description: CRC checksum
=============================================================================*/
__device__  uint32_t singletable_crc32c_ui32(uint32_t ui32_crc, uint32_t ui32_data)
{
	ui32_to_ui8_t u;
	u.ui32 = ui32_data;

	/* 4 bytes*/
	ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[0u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[1u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[2u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[3u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	return ui32_crc;
}


/* ==========================================================================
  Description: Fletcher checksum
=============================================================================*/
__device__ uint32_t Fletcher32c_ui32(uint32_t Prev_Fletcher, uint32_t ui32_data)
{
	  ui32_to_ui16_t v;
    ui32_to_ui16_t Fletcher;
    v.ui32        = ui32_data;
    Fletcher.ui32 = Prev_Fletcher;

	Fletcher.ui16[0] += v.ui16[0];
	Fletcher.ui16[1] += Fletcher.ui16[0];
	Fletcher.ui16[0] += v.ui16[1];
	Fletcher.ui16[1] += Fletcher.ui16[0];
	Fletcher.ui16[0] %= 255;
	Fletcher.ui16[1] %= 255;

	return Fletcher.ui32;
}

// atomicXor(&ES_a[0], (uint32_t) *((uint32_t*) &a[0]));

/* Code Included to test the function __add32
    uint32_t val_a = 4294967295;
    uint32_t val_b = 1;
    uint32_t val_c;
    val_c = 0;
    printf("Before a: %u b: %u c: %u\n",val_a,val_b,val_c);
    val_c = __add32( val_a, val_b);
    printf("Ones a: %u b: %u c: %u\n",val_a,val_b,val_c);
*/