#include "hip/hip_runtime.h"

/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS GEMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This is kernel computes
  the general matrix product (GEMM) using single-precision floating-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 128x128x8 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SGEMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

  This example has delibrately been kept similar to the basic_gemm example from cutass-1.3 to 
  highlight the minimum amount of differences needed to transition to cutlass-2.0.

  Cutlass-1.3 sgemm: https://github.com/NVIDIA/cutlass/blob/master/examples/00_basic_gemm/basic_gemm.cu
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
// Helper methods to check for errors
#include "helper.h"
//
#include <sched.h>
#include <string.h>

#ifndef NAME
  #define NAME "NO_NAME"
#endif

#ifndef CHAR_BIT
  #define CHAR_BIT 8u
#endif

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/gemm.h"
#include <time.h>

// lock memory
#include <sys/mman.h>

#define PUT_IN_REGISTER								 /* dummy definition  for Windows 32 */

#ifndef TIMING_EXP
  #define TIMING_EXP 0u
#endif

#ifndef DC_EXP
  #define DC_EXP 0u
#endif 

#ifndef ES_EXP
  #define ES_EXP 0u
#endif 


typedef float    float32_t;
typedef union ui64_to_ui32 {
	uint64_t ui64;
	uint32_t ui32[2];
} ui64_to_ui32_t;


typedef float    float32_t;
typedef double   float64_t;
typedef void     void_t;

static void_t matrix2rand(float32_t * paf32_matrix, uint32_t ui32_max_rows, uint32_t ui32_max_columns)
{
	uint32_t ui32_idx;

	for (ui32_idx = 0u; ui32_idx < (ui32_max_rows * ui32_max_columns); ui32_idx++)
	{
		*paf32_matrix++ = (float32_t)rand();
	}
}

//
// CUTLASS includes needed for single-precision GEMM kernel
//


// Include Smmm.h (MMM employed in the previous paper)
//#include "cutlass/sequential_MMM/Smmm.h"

  // Definition of an struct to store th values of the Execution Signatures
struct ESs{
    uint32_t A;
    uint32_t B;
    uint32_t C;
  };


#define DEF_TIME_VAR(t) clock_t t;
#define GET_TIME(t) t = clock();
#define GET_TIME_DIFF(tmr_start, tmr_end, f_time_interval) f_time_interval = ((tmr_end - tmr_start) > 0) ? ( (float32_t)  (tmr_end - tmr_start) / CLOCKS_PER_SEC) : ( (float32_t)  (tmr_start - tmr_end)/ CLOCKS_PER_SEC)

#define TIME_MEASUREMENT_LOOPS  200u
#define INITIAL_TIME_MEASUREMENT 10u
#define TIME_SEC2USEC       ((uint32_t) 1000000u) /*!< Microseconds per second*/

#if (INTERNAL_ES==CRC_CHECKSUM) || (INTERMEDIATE_ES==CRC_CHECKSUM) || (CRC_CHECKSUM==EXTERNAL_ES)
    #define CRC_table_elements 256u
    /* ==============================================================================================================
    * 										CONSTS
    * ============================================================================================================== */
    uint32_t kaui32_crc_table[CRC_table_elements] = 
    {
      0x00000000L, 0xF26B8303L, 0xE13B70F7L, 0x1350F3F4L,
      0xC79A971FL, 0x35F1141CL, 0x26A1E7E8L, 0xD4CA64EBL,
      0x8AD958CFL, 0x78B2DBCCL, 0x6BE22838L, 0x9989AB3BL,
      0x4D43CFD0L, 0xBF284CD3L, 0xAC78BF27L, 0x5E133C24L,
      0x105EC76FL, 0xE235446CL, 0xF165B798L, 0x030E349BL,
      0xD7C45070L, 0x25AFD373L, 0x36FF2087L, 0xC494A384L,
      0x9A879FA0L, 0x68EC1CA3L, 0x7BBCEF57L, 0x89D76C54L,
      0x5D1D08BFL, 0xAF768BBCL, 0xBC267848L, 0x4E4DFB4BL,
      0x20BD8EDEL, 0xD2D60DDDL, 0xC186FE29L, 0x33ED7D2AL,
      0xE72719C1L, 0x154C9AC2L, 0x061C6936L, 0xF477EA35L,
      0xAA64D611L, 0x580F5512L, 0x4B5FA6E6L, 0xB93425E5L,
      0x6DFE410EL, 0x9F95C20DL, 0x8CC531F9L, 0x7EAEB2FAL,
      0x30E349B1L, 0xC288CAB2L, 0xD1D83946L, 0x23B3BA45L,
      0xF779DEAEL, 0x05125DADL, 0x1642AE59L, 0xE4292D5AL,
      0xBA3A117EL, 0x4851927DL, 0x5B016189L, 0xA96AE28AL,
      0x7DA08661L, 0x8FCB0562L, 0x9C9BF696L, 0x6EF07595L,
      0x417B1DBCL, 0xB3109EBFL, 0xA0406D4BL, 0x522BEE48L,
      0x86E18AA3L, 0x748A09A0L, 0x67DAFA54L, 0x95B17957L,
      0xCBA24573L, 0x39C9C670L, 0x2A993584L, 0xD8F2B687L,
      0x0C38D26CL, 0xFE53516FL, 0xED03A29BL, 0x1F682198L,
      0x5125DAD3L, 0xA34E59D0L, 0xB01EAA24L, 0x42752927L,
      0x96BF4DCCL, 0x64D4CECFL, 0x77843D3BL, 0x85EFBE38L,
      0xDBFC821CL, 0x2997011FL, 0x3AC7F2EBL, 0xC8AC71E8L,
      0x1C661503L, 0xEE0D9600L, 0xFD5D65F4L, 0x0F36E6F7L,
      0x61C69362L, 0x93AD1061L, 0x80FDE395L, 0x72966096L,
      0xA65C047DL, 0x5437877EL, 0x4767748AL, 0xB50CF789L,
      0xEB1FCBADL, 0x197448AEL, 0x0A24BB5AL, 0xF84F3859L,
      0x2C855CB2L, 0xDEEEDFB1L, 0xCDBE2C45L, 0x3FD5AF46L,
      0x7198540DL, 0x83F3D70EL, 0x90A324FAL, 0x62C8A7F9L,
      0xB602C312L, 0x44694011L, 0x5739B3E5L, 0xA55230E6L,
      0xFB410CC2L, 0x092A8FC1L, 0x1A7A7C35L, 0xE811FF36L,
      0x3CDB9BDDL, 0xCEB018DEL, 0xDDE0EB2AL, 0x2F8B6829L,
      0x82F63B78L, 0x709DB87BL, 0x63CD4B8FL, 0x91A6C88CL,
      0x456CAC67L, 0xB7072F64L, 0xA457DC90L, 0x563C5F93L,
      0x082F63B7L, 0xFA44E0B4L, 0xE9141340L, 0x1B7F9043L,
      0xCFB5F4A8L, 0x3DDE77ABL, 0x2E8E845FL, 0xDCE5075CL,
      0x92A8FC17L, 0x60C37F14L, 0x73938CE0L, 0x81F80FE3L,
      0x55326B08L, 0xA759E80BL, 0xB4091BFFL, 0x466298FCL,
      0x1871A4D8L, 0xEA1A27DBL, 0xF94AD42FL, 0x0B21572CL,
      0xDFEB33C7L, 0x2D80B0C4L, 0x3ED04330L, 0xCCBBC033L,
      0xA24BB5A6L, 0x502036A5L, 0x4370C551L, 0xB11B4652L,
      0x65D122B9L, 0x97BAA1BAL, 0x84EA524EL, 0x7681D14DL,
      0x2892ED69L, 0xDAF96E6AL, 0xC9A99D9EL, 0x3BC21E9DL,
      0xEF087A76L, 0x1D63F975L, 0x0E330A81L, 0xFC588982L,
      0xB21572C9L, 0x407EF1CAL, 0x532E023EL, 0xA145813DL,
      0x758FE5D6L, 0x87E466D5L, 0x94B49521L, 0x66DF1622L,
      0x38CC2A06L, 0xCAA7A905L, 0xD9F75AF1L, 0x2B9CD9F2L,
      0xFF56BD19L, 0x0D3D3E1AL, 0x1E6DCDEEL, 0xEC064EEDL,
      0xC38D26C4L, 0x31E6A5C7L, 0x22B65633L, 0xD0DDD530L,
      0x0417B1DBL, 0xF67C32D8L, 0xE52CC12CL, 0x1747422FL,
      0x49547E0BL, 0xBB3FFD08L, 0xA86F0EFCL, 0x5A048DFFL,
      0x8ECEE914L, 0x7CA56A17L, 0x6FF599E3L, 0x9D9E1AE0L,
      0xD3D3E1ABL, 0x21B862A8L, 0x32E8915CL, 0xC083125FL,
      0x144976B4L, 0xE622F5B7L, 0xF5720643L, 0x07198540L,
      0x590AB964L, 0xAB613A67L, 0xB831C993L, 0x4A5A4A90L,
      0x9E902E7BL, 0x6CFBAD78L, 0x7FAB5E8CL, 0x8DC0DD8FL,
      0xE330A81AL, 0x115B2B19L, 0x020BD8EDL, 0xF0605BEEL,
      0x24AA3F05L, 0xD6C1BC06L, 0xC5914FF2L, 0x37FACCF1L,
      0x69E9F0D5L, 0x9B8273D6L, 0x88D28022L, 0x7AB90321L,
      0xAE7367CAL, 0x5C18E4C9L, 0x4F48173DL, 0xBD23943EL,
      0xF36E6F75L, 0x0105EC76L, 0x12551F82L, 0xE03E9C81L,
      0x34F4F86AL, 0xC69F7B69L, 0xD5CF889DL, 0x27A40B9EL,
      0x79B737BAL, 0x8BDCB4B9L, 0x988C474DL, 0x6AE7C44EL,
      0xBE2DA0A5L, 0x4C4623A6L, 0x5F16D052L, 0xAD7D5351L
    };
    __constant__ uint32_t d_CRC_table_constant[CRC_table_elements];
    __shared__ uint32_t d_CRC_table_shared[CRC_table_elements];


/* ==========================================================================
  Description: CRC function
=============================================================================*/
__host__ uint32_t singletable_crc32c_ui32(uint32_t ui32_crc, uint32_t ui32_data)
{
	ui32_to_ui8_t u;
	u.ui32 = ui32_data;

	/* 4 bytes*/
  uint32_t prev_ui32_crc = ui32_crc;
	ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[0u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[1u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[2u]) & 0x00ffu] ^ (ui32_crc >> 8u);
	ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[3u]) & 0x00ffu] ^ (ui32_crc >> 8u);
  return ui32_crc;
}
#endif

/* ==========================================================================
  Description: Bit inversion function
=============================================================================*/
static void_t mem_fi(float32_t* const paf32_m, uint32_t ui32_bit_idx)
{
	uint32_t ui32_idx_flt = ui32_bit_idx / (sizeof(float32_t)*CHAR_BIT),
		ui32_idx_flt_bit = ui32_bit_idx % (sizeof(float32_t)*CHAR_BIT);
	uint32_t ui32_f_d = *((uint32_t *)&paf32_m[ui32_idx_flt]);

	ui32_f_d ^= (1u << ui32_idx_flt_bit);
	paf32_m[ui32_idx_flt] = *((float32_t *)&ui32_f_d);
}

// Definition of the sequential MMM (not required. It belongs to a test that try to compare sequential ES and parallel ES)
 ESs smm_xor_internal(uint32_t ui32_m, uint32_t ui32_n, uint32_t ui32_k, float32_t f32_alpha,  float32_t*  paf32_ma,  float32_t*  paf32_mb, float32_t *paf32_mc)
{
	uint32_t ui32_idx_i = 0u,
		ui32_idx_j = 0u,
		ui32_idx_k = 0u,
		ui32_idx_a = 0u,
		ui32_idx_b = 0u,
		ui32_idx_c = 0u,
		ui32_idx_b_ref = 0u,
		ui32_idx_c_ref = 0u;

	float32_t f32_a_part = 0.0f,
		f32_b = 0.0f,
		f32_c = 0.0f;

	/* XOR checksum */
  struct ESs ES;
  ES.A = 0u;
  ES.B = 0u;
  ES.C = 0u;


	// Verification of the input values
	assert(paf32_ma != NULL);
	assert(paf32_mb != NULL);
	assert(paf32_mc != NULL);

	for (ui32_idx_i = 0u; ui32_idx_i < ui32_m; ui32_idx_i++)
	{
		ui32_idx_b_ref = 0u;
		for (ui32_idx_k = 0u; ui32_idx_k < ui32_k; ui32_idx_k++, ui32_idx_a++)
		{
			PUT_IN_REGISTER f32_a_part = f32_alpha * paf32_ma[ui32_idx_a];
			ES.A ^= (uint32_t) *((uint32_t*)&f32_a_part);

			for (ui32_idx_j = 0u, ui32_idx_b = ui32_idx_b_ref, ui32_idx_c = ui32_idx_c_ref; ui32_idx_j < ui32_n; ui32_idx_j++, ui32_idx_b++, ui32_idx_c++)
			{
				f32_b = paf32_mb[ui32_idx_b];
				paf32_mc[ui32_idx_c] += f32_a_part * f32_b;
				f32_c = paf32_mc[ui32_idx_c];

				/* XOR checksum */
				ES.B ^= (uint32_t) *((uint32_t*)&f32_b);
        //printf("Value[%u]=%u\n",ui32_idx_b,(uint32_t) *((uint32_t*)&f32_b));
				ES.C ^= (uint32_t) *((uint32_t*)&f32_c);
			}
      //printf("%u\n",ES.B);
			ui32_idx_b_ref += ui32_n;
		}
		ui32_idx_c_ref += ui32_n;
	}
	//ui32_xor = (ui32_xor_a ^ ui32_xor_b) ^ ui32_xor_c;
	return ES;
}


/*==============================================================================================================
**									Name: smm_ones_internal
==============================================================================================================*/
/*!
** @brief Matrix-matrix multiplication (MMM) with One's checksum in the internal loop
**
** @param[in] ui32_m 		Number of matrix A rows 								[0…ui32_m]
** @param[in] ui32_n 		Number of matrix B columns 								[0…ui32_n]
** @param[in] ui32_k 		Number of matrix A columns / Number of matrix B rows 	[0…ui32_k]
** @param[in] f32_alpha Correction factor
** @param[in] paf32_ma 	Pointer to the first position of an array of floats (A matrix direction)
** @param[in] paf32_mb 	Pointer to the first position of an array of floats (B matrix direction)
** @param[in] paf32_mc 	Pointer to the first position of an array of floats (B matrix direction)
**
** @return uint32_t  	ui32_xor	Return the Execution signature of the MMM
==============================================================================================================*/
ESs smm_ones_internal(uint32_t ui32_m, uint32_t ui32_n, uint32_t ui32_k, float32_t f32_alpha,  float32_t*  paf32_ma,  float32_t*  paf32_mb, float32_t*  paf32_mc)
{
	uint32_t ui32_idx_i = 0u,
		ui32_idx_j = 0u,
		ui32_idx_k = 0u,
		ui32_idx_a = 0u,
		ui32_idx_b = 0u,
		ui32_idx_c = 0u,
		ui32_idx_b_ref = 0u,
		ui32_idx_c_ref = 0u;

	float32_t f32_a_part = 0.0f,
		f32_b = 0.0f,
		f32_c = 0.0f;

	/* One's complement checksum */
	ui64_to_ui32_t Ones_Checksum_a,
		Ones_Checksum_b,
		Ones_Checksum_c,
		Ones_Checksum;

	Ones_Checksum_a.ui64 = 0u;
	Ones_Checksum_b.ui64 = 0u;
	Ones_Checksum_c.ui64 = 0u;


	// Verification of the input values
	assert(paf32_ma != NULL);
	assert(paf32_mb != NULL);
	assert(paf32_mc != NULL);

	for (ui32_idx_i = 0u; ui32_idx_i < ui32_m; ui32_idx_i++)
	{
		ui32_idx_b_ref = 0u;
		for (ui32_idx_k = 0u; ui32_idx_k < ui32_k; ui32_idx_k++, ui32_idx_a++)
		{
			PUT_IN_REGISTER f32_a_part = f32_alpha * paf32_ma[ui32_idx_a];
			Ones_Checksum_a.ui64 += (uint64_t) * ((uint32_t*)&f32_a_part);
			Ones_Checksum_a.ui32[0] += Ones_Checksum_a.ui32[1];
			Ones_Checksum_a.ui32[0] = ~Ones_Checksum_a.ui32[0];
      Ones_Checksum_a.ui32[1] = 0;

			for (ui32_idx_j = 0u, ui32_idx_b = ui32_idx_b_ref, ui32_idx_c = ui32_idx_c_ref; ui32_idx_j < ui32_n;
				ui32_idx_j++, ui32_idx_b++, ui32_idx_c++)
			{
				f32_b = paf32_mb[ui32_idx_b];
				paf32_mc[ui32_idx_c] += f32_a_part * f32_b;
				f32_c = paf32_mc[ui32_idx_c];
				/* One's complement checksum */
				Ones_Checksum_b.ui64 += (uint64_t) * ((uint32_t*)&f32_b);
				Ones_Checksum_b.ui32[0] += Ones_Checksum_b.ui32[1];
				Ones_Checksum_b.ui32[0] = ~Ones_Checksum_b.ui32[0];

				Ones_Checksum_c.ui64 += (uint64_t) * ((uint32_t*)&f32_c);
				Ones_Checksum_c.ui32[0] += Ones_Checksum_c.ui32[1];
				Ones_Checksum_c.ui32[0] = ~Ones_Checksum_c.ui32[0];

        Ones_Checksum_b.ui32[1] = 0;
        Ones_Checksum_c.ui32[1] = 0;
			}
			ui32_idx_b_ref += ui32_n;
		}
		ui32_idx_c_ref += ui32_n;
	}
  /*
	Ones_Checksum.ui64 = (Ones_Checksum_a.ui64 + Ones_Checksum_b.ui64);
	Ones_Checksum.ui32[0] += Ones_Checksum.ui32[1];
	Ones_Checksum.ui32[0] = ~Ones_Checksum.ui32[0];
	Ones_Checksum.ui64 += Ones_Checksum_c.ui64;
	Ones_Checksum.ui32[0] += Ones_Checksum.ui32[1];
	Ones_Checksum.ui32[0] = ~Ones_Checksum.ui32[0];
  */
   struct ESs ES;
  ES.A = Ones_Checksum_a.ui32[0];
  ES.B = Ones_Checksum_b.ui32[0];
  ES.C = Ones_Checksum_c.ui32[0];

	return ES;
}


///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc,
  uint32_t *d_ES_a,
  uint32_t *d_ES_b,
  uint32_t *d_ES_c) {

  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h`

  using ColumnMajor = cutlass::layout::ColumnMajor;

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                  ColumnMajor,  // Layout of A matrix
                                                  float,        // Data-type of B matrix
                                                  ColumnMajor,  // Layout of B matrix
                                                  float,        // Data-type of C matrix
                                                  ColumnMajor>; // Layout of C matrix

  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  //printf("\n Direction of h_ES_0: %p and value: %f \n", (void *) h_ES_0, h_ES_0[4]);

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta},// Scalars used in the Epilogue
                              d_ES_a,   // Pointer to d_ES_a
                              d_ES_b,   // Pointer to d_ES_b
                              d_ES_c);   // Pointer to d_ES_c
                             

  // Code included by JFdez: I have to include in args variable this: d_ES_0 and d_ES_1

  //
  // Launch the CUTLASS GEMM kernel.
  //
  
  cutlass::Status status = gemm_operator(args);

  //
  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}


/// Naive reference GEMM computation.
__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    float accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }
    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

/// Reference GEMM computation.
hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;



  /*==============================================================================
  //   1)                  CPU affinity
	==============================================================================*/
	cpu_set_t mask;
	CPU_ZERO(&mask);
	CPU_SET(0, &mask);
	if(sched_setaffinity(0, sizeof(cpu_set_t), &mask) < 0)
	{
		perror("sched_setaffinity failed");
		exit(-1);
	}
  
  /*==============================================================================
	//   2)               Real-time priority
  ==============================================================================*/
	struct sched_param param;
	param.sched_priority = 98;
	if(sched_setscheduler(0, SCHED_FIFO, &param) < 0)
	{
		perror("sched_setscheduler failed");
		exit(-1);
	}
  
    /*==============================================================================
	//   3)                  Memory locking
	==============================================================================*/
	if(mlockall(MCL_CURRENT | MCL_FUTURE) < 0)
	{
		perror("mlockall failed");
		exit(-1);
	} 

  /*==============================================================================
  //                       Timing variables
  ==============================================================================*/
  struct timespec begin, end;
  uint64_t time_max_ns = 0;
  long long iteration_max_time = 0;
  uint64_t time_ns;

  /*==============================================================================
  //                      Statistical variables
  ==============================================================================*/

	double M_time = 0.0;
	double S_time = 0.0;

  DEF_TIME_VAR(tmr_start);
	DEF_TIME_VAR(tmr_end);
  float64_t time_interval;

    // Compute leading dimensions for each matrix.
    int lda = M;
    int ldb = K;
    int ldc = M;

    // Compute size in bytes of the C matrix.
    size_t sizeof_C = sizeof(float) * ldc * N;

  // =============================================================
  // Author:  Javier Fdez
  // Date:    2021/08/17
  // Summary: In the following chunk of code the variables ES_0 and  
  //          ES_1are initialized in CPU and GPU. In these variables 
  //          are stored the Execution Signatures (1 per thread*SMP)
  // =============================================================
  //  Definition of an struct with the values of the ES:
  struct ESs h_ES;
  struct ESs d_ES;

  // =============================================================
  //  4) Define the pointers to ES_a,b,c
  // =============================================================
  uint32_t nElem_ES = 256;
  size_t nBytes_ES = nElem_ES * sizeof(uint32_t);

  // Define pointers to ES_a, ES_b and ES_c in CPU (host)
  uint32_t *h_ES_a;
  uint32_t *h_ES_b;
  uint32_t *h_ES_c;
  #if (INTERNAL_ES==CRC_CHECKSUM) || (INTERMEDIATE_ES==CRC_CHECKSUM) || (CRC_CHECKSUM==EXTERNAL_ES)
    uint32_t *h_CRC_table;
  #endif

  // Allocate ES_a, ES_b and ES_c in CPU 
  h_ES_a = (uint32_t *) malloc(nBytes_ES);
  h_ES_b = (uint32_t *) malloc(nBytes_ES);
  h_ES_c = (uint32_t *) malloc(nBytes_ES);

  // Initialice to 0 all values of ES_a, ES_b and ES_c
  memset(h_ES_a,0,nBytes_ES);
  memset(h_ES_b,0,nBytes_ES);
  memset(h_ES_c,0,nBytes_ES);
  
  // =============================================================
  //   5) Define the pointers to ES_a_ref,b,c
  // =============================================================
  // Define pointers to ES_a_ref,b,c in CPU (host)
  uint32_t *h_ES_a_ref;
  uint32_t *h_ES_b_ref;
  uint32_t *h_ES_c_ref;

   // Allocate ES_a_ref, ES_b_ref and ES_c_ref in CPU 
  h_ES_a_ref = (uint32_t *) malloc(nBytes_ES);
  h_ES_b_ref = (uint32_t *) malloc(nBytes_ES);
  h_ES_c_ref = (uint32_t *) malloc(nBytes_ES);
  

  // Initialice to 0 all values of ES_a_ref, ES_b_ref and ES_c_ref
  memset(h_ES_a_ref,0,nBytes_ES);
  memset(h_ES_b_ref,0,nBytes_ES);
  memset(h_ES_c_ref,0,nBytes_ES);
  

  // =============================================================
  //   6) Define the pointers to d_ES_a,b,c
  // =============================================================
  // Define pointers to d_ES_a,b,c and CRC lookup table in GPU (device)
  uint32_t *d_ES_a;
  uint32_t *d_ES_b;
  uint32_t *d_ES_c;

  // Allocate d_ES_a,b,c in GPU
  result = hipMalloc((uint32_t **) &d_ES_a, nBytes_ES);
  if (result != hipSuccess) {
    std::cerr << "Failed to Allocate d_ES_a: "<< hipGetErrorString(result) << std::endl;
    hipFree(d_ES_a);
    return result;
  }
  result = hipMalloc((uint32_t **) &d_ES_b, nBytes_ES);
  if (result != hipSuccess) {
    std::cerr << "Failed to Allocate d_ES_b: "<< hipGetErrorString(result) << std::endl;
    hipFree(d_ES_a);
    hipFree(d_ES_b);
    return result;
  }
  result = hipMalloc((uint32_t **) &d_ES_c, nBytes_ES);
  if (result != hipSuccess) {
    std::cerr << "Failed to Allocate d_ES_c: "<< hipGetErrorString(result) << std::endl;
    hipFree(d_ES_a);
    hipFree(d_ES_b);
    hipFree(d_ES_c);
    return result;
  }

  // =============================================================
  //   Transfer h_ES_a... -> d_ES_a... (from host to device)
  // =============================================================
  result = hipMemcpy(d_ES_a, h_ES_a, nBytes_ES, hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_ES_a matrix to d_ES_a: " << hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        hipFree(d_ES_c);
        return result;
    }

    // Initialize d_ES_b to 0
    result = hipMemcpy(d_ES_b, h_ES_b, nBytes_ES, hipMemcpyHostToDevice);
    if (result != hipSuccess) 
    {
        std::cerr << "Failed to copy h_ES_b matrix to h_ES_b: " << hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        hipFree(d_ES_c);
        return result;
    }

    // Initialize d_ES_c to 0
    hipMemcpy(d_ES_c, h_ES_c, nBytes_ES, hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_ES_c matrix to d_ES_c " << hipGetErrorString(result) << std::endl;
        hipFree(d_ES_a);
        hipFree(d_ES_b);
        hipFree(d_ES_c);
        return result;
    }

#if (INTERNAL_ES==CRC_CHECKSUM) || (INTERMEDIATE_ES==CRC_CHECKSUM) || (CRC_CHECKSUM==EXTERNAL_ES)
  // =============================================================
  //  7) Copy the CRC lookup table from host to device
  // =============================================================
  result = hipMemcpyToSymbol(HIP_SYMBOL(d_CRC_table_constant), kaui32_crc_table, CRC_table_elements*sizeof(uint32_t));
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate Constant Memory: "
      << hipGetErrorString(result) << std::endl;
      hipFree(d_ES_a);
      hipFree(d_ES_b);
      hipFree(d_ES_c);
      hipFree(d_CRC_table_constant);
    }
#endif


  // ==============================================================================
  // Brief: Initialization of A, B, C_reference and C_cutlass. 
  // Hypothesis: the values of A and B follow a pattern in the initialization if it 
  // is done in with the GPU. This sequential implementation aid to avoid this.
  //===============================================================================
  uint32_t nBytes_a = M * K * sizeof(float);
  uint32_t nBytes_b = K * N * sizeof(float);
  uint32_t nBytes_c = M * N * sizeof(float);

  // ==============================================================================
  // 8) Define pointers to h_a, h_b and h_c in CPU (host)
  // ==============================================================================
  float *h_a;
  float *h_b;
  float *h_c;

  // Allocate h_a, h_b and h_c in CPU 
  h_a     = (float *) malloc(nBytes_a);
  h_b     = (float *) malloc(nBytes_b);
  h_c     = (float *) malloc(nBytes_c);

  // Initialice to 0 all values of h_c
  memset(h_c,0,nBytes_c);
  
  // Initialization of the values of h_a and h_b
  #if (1==ES_EXP)
      matrix2rand(h_a,M,K);
      matrix2rand(h_b,K,N);
  #elif (1==TIMING_EXP)
      matrix2rand(h_a,M,K);
      matrix2rand(h_b,K,N);
  #elif (1==DC_EXP) 
      memset(h_a,0,nBytes_a);
      memset(h_b,0,nBytes_b);
  #endif


  // ==============================================================================
  //   8) Define pointers to matrices in GPU device memory, allocate, initialize.
  // ==============================================================================
  float *A;
  float *B;
  float *C_cutlass;
  float *C_reference;

  // Allocate memory on GPU
  result =  hipMalloc(&A, nBytes_a);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate A: "
      << hipGetErrorString(result) << std::endl;}

  result =  hipMalloc(&B, nBytes_b);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate B: "
      << hipGetErrorString(result) << std::endl;}

  result =  hipMalloc(&C_cutlass, nBytes_c);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate C_cutlass: "
      << hipGetErrorString(result) << std::endl;}

  result =  hipMalloc(&C_reference, nBytes_c);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate C_reference: "
      << hipGetErrorString(result) << std::endl;}
  
  // Copy the values initialized and stored in host to the device (h_a -> A, h_b -> B ...)
  result = hipMemcpy (A,h_a,nBytes_a,hipMemcpyHostToDevice);
 if (result != hipSuccess) {
    std::cerr << "Failed to copy h_a matrix to A: " << hipGetErrorString(result) << std::endl;
    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);
    return result;
  }

  result = hipMemcpy (B,h_b,nBytes_b,hipMemcpyHostToDevice);
 if (result != hipSuccess) {
    std::cerr << "Failed to copy h_b matrix to B: " << hipGetErrorString(result) << std::endl;
    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);
    return result;
  }

  result = hipMemcpy (C_cutlass,h_c,nBytes_c,hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: " << hipGetErrorString(result) << std::endl;
    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);
    return result;
  }

  result = hipMemcpy (C_reference,h_c,nBytes_c,hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: " << hipGetErrorString(result) << std::endl;
    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);
    return result;
  }
  
  /* ==============================================================================
    Brief: Diagnostic Coverage experiments
  ==============================================================================*/
#if (1==DC_EXP)
  //size_t size_a,size_b;
	uint32_t ui32_idx_bit,
		ui32_dc_cnt_all,
		ui32_dc_cnt_a,
    ui32_dc_cnt_b;
	float32_t f32_alpha = 1.0f;

	DEF_TIME_VAR(tmr_start_exp);
	DEF_TIME_VAR(tmr_end_exp);
  printf("===============================\nDIAGNOSTIC COVERAGE EXPERIMENT\n===============================\n");

#if (ES_EXP==1)
  #define size_a  100u
  #define size_b  0u
#else
	#define size_a  (DIM_M * DIM_K * sizeof(float32_t) * CHAR_BIT)
	#define size_b  (DIM_N * DIM_K * sizeof(float32_t) * CHAR_BIT)
#endif

  // ==============================================================================
  //  0)             DC Set-up environment
  // ==============================================================================
	// Fault injection campaing 
	//uint32_t ui32_comb_a_max = size_a;
  uint32_t ui32_comb_a_max = size_a;
	uint32_t ui32_comb_b_max = size_b;

  ui32_dc_cnt_a = 0u;
  ui32_dc_cnt_b = 0u;
  ui32_dc_cnt_all = 0u;
  uint32_t ui32_idx_bit_aux = 0u;

  #define ui32_20_percent_a size_a*20/100
  #define ui32_40_percent_a size_a*40/100
  #define ui32_60_percent_a size_a*60/100
  #define ui32_80_percent_a size_a*80/100
  

  // =======================================================================================
  // 1) Store the ES_a,b and c that will be employed as reference
  // =======================================================================================
  // Launch the kernel to GPU and verify that has not return an error
    result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_reference, ldc, d_ES_a, d_ES_b, d_ES_c);
    hipDeviceSynchronize();

    if (result != hipSuccess) 
    {
        std::cerr << "CUTLASS GEMM kernel failed: " << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
    }
    // Copy to host the values of the ES of A, B and C performed and stored in the GPU device (h_ES_a_ref)
    result = hipMemcpy(h_ES_a_ref, d_ES_a, nBytes_ES, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_a matrix to h_ES_a (return)" << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }

    result = hipMemcpy(h_ES_b_ref, d_ES_b, nBytes_ES, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_b matrix to h_ES_b (return)" << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }

    result = hipMemcpy(h_ES_c_ref, d_ES_c, nBytes_ES, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_c matrix to h_ES_c (return)" << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }
  

  for (ui32_idx_bit = ui32_idx_bit_aux; ui32_idx_bit < ui32_comb_a_max; ui32_idx_bit++) 
  {
    // =======================================================================================
    // Modify a bit of the matrix A
    // =======================================================================================
    #if (ES_EXP==0)
      mem_fi(&h_a[0], ui32_idx_bit);
    #endif

    // =======================================================================================
    // Copy the values initialized and stored in host to the device (h_a -> A, h_b -> B ...)
    // =======================================================================================
    result = hipMemcpy (A,h_a,nBytes_a,hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_a matrix to A: "
          << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
      }

    /*result = hipMemcpy (B,h_b,nBytes_b,hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_b matrix to B: "
          << hipGetErrorString(result) << std::endl;

        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);

        return result;
    }
    */

    result = hipMemcpy (C_cutlass,h_c,nBytes_c,hipMemcpyHostToDevice);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy C_cutlass matrix to C_reference: " << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }

    // =======================================================================================
    // 1) Initialize h_ES_a,b,c 
    // 2) Re-initialize d_ES_a,b,c with zero values
    // =======================================================================================
    // Initialice to 0 all values of h_ES_a,b,c
    memset(h_ES_a,0,nBytes_ES);
    memset(h_ES_b,0,nBytes_ES);
    memset(h_ES_c,0,nBytes_ES);

    result = hipMemcpy(d_ES_a, h_ES_a, nBytes_ES, hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_ES_a matrix to d_ES_a: " << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
    }

    // Re-initialize d_ES_b
    result = hipMemcpy(d_ES_b, h_ES_b, nBytes_ES, hipMemcpyHostToDevice);
    if (result != hipSuccess) 
    {
        std::cerr << "Failed to copy h_ES_b matrix to h_ES_b: " << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
    }

    // Re-initialize d_ES_c
    hipMemcpy(d_ES_c, h_ES_c, nBytes_ES, hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        std::cerr << "Failed to copy h_ES_c matrix to d_ES_c " << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
    }

    // =======================================================================================
    // Launch the kernel to GPU and verify that has not return an error
    // =======================================================================================
    result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, d_ES_a, d_ES_b, d_ES_c);
    hipDeviceSynchronize();

    if (result != hipSuccess) 
    {
        std::cerr << "CUTLASS GEMM kernel failed: " << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
    }

    // =======================================================================================
    // Launch the kernel to GPU and verify that has not return an error
    // =======================================================================================
    // Copy to host the values of the ES of A, B and C performed and stored in the GPU device
    result = hipMemcpy(h_ES_a, d_ES_a, nBytes_ES, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_a matrix to h_ES_a (return)" << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }

    result = hipMemcpy(h_ES_b, d_ES_b, nBytes_ES, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_b matrix to h_ES_b (return)" << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }

    result = hipMemcpy(h_ES_c, d_ES_c, nBytes_ES, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
      std::cerr << "Failed to copy d_ES_c matrix to h_ES_c (return)" << hipGetErrorString(result) << std::endl;
      hipFree(C_reference);
      hipFree(C_cutlass);
      hipFree(B);
      hipFree(A);
      return result;
    }

    // =======================================================================================
    // Verify that the ES_a,b,c is the same than the values stored in ES_a_ref,b,c
    // =======================================================================================
    if ((memcmp(h_ES_a,h_ES_a_ref,nBytes_ES)!=0) || (memcmp(h_ES_b,h_ES_b_ref,nBytes_ES)!=0) || (memcmp(h_ES_c,h_ES_c_ref,nBytes_ES)!=0)) {
          ui32_dc_cnt_a += 1u;
    }
    ui32_dc_cnt_all += 1u;

    // =======================================================================================
    // TO flip again the bit that was fliped in host
    // =======================================================================================
    #if (ES_EXP==0)
      mem_fi(&h_a[0], ui32_idx_bit);
    #endif

    // =======================================================================================
    // dummy way of now of the process is being 
    // =======================================================================================
    /*switch(ui32_idx_bit) 
    {
      case ui32_20_percent_a:
          printf("Processing: [\t20%%\t");
          break;
      case ui32_40_percent_a:
          printf("=>40%%\t");
          break;
      case ui32_60_percent_a:
          printf("=>60%%\t");
          break;
      case ui32_80_percent_a:
          printf("=>80%%\n");
          break;
    }*/
  }
  printf("Number of executions: %d\nDifferents ES: %d\n",ui32_comb_a_max, ui32_dc_cnt_a);






#if (ES_EXP==0)
  // =======================================================================================
  // Copy the values initialized and stored in host to the device (h_a -> A)
  // =======================================================================================
  result = hipMemcpy (A,h_a,nBytes_a,hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "Failed to copy h_a matrix to A: " << hipGetErrorString(result) << std::endl;
    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);
    return result;
  }

  for (ui32_idx_bit = ui32_idx_bit_aux; ui32_idx_bit < ui32_comb_b_max; ui32_idx_bit++) 
    {
      // =======================================================================================
      // Modify a bit of the matrix B
      // =======================================================================================
      mem_fi(&h_b[0], ui32_idx_bit);

      result = hipMemcpy (B,h_b,nBytes_b,hipMemcpyHostToDevice);
      if (result != hipSuccess) {
          std::cerr << "Failed to copy h_b matrix to B: " << hipGetErrorString(result) << std::endl;
          hipFree(C_reference);
          hipFree(C_cutlass);
          hipFree(B);
          hipFree(A);
          return result;
      }
      

      result = hipMemcpy (C_cutlass,h_c,nBytes_c,hipMemcpyHostToDevice);
      if (result != hipSuccess) {
        std::cerr << "Failed to copy C_cutlass matrix to C_reference: " << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
      }

      // =======================================================================================
      // 1) Initialize h_ES_a,b,c 
      // 2) Re-initialize d_ES_a,b,c with zero values
      // =======================================================================================
      // Initialice to 0 all values of h_ES_a,b,c
      memset(h_ES_a,0,nBytes_ES);
      memset(h_ES_b,0,nBytes_ES);
      memset(h_ES_c,0,nBytes_ES);

      result = hipMemcpy(d_ES_a, h_ES_a, nBytes_ES, hipMemcpyHostToDevice);
      if (result != hipSuccess) {
          std::cerr << "Failed to copy h_ES_a matrix to d_ES_a: " << hipGetErrorString(result) << std::endl;
          hipFree(C_reference);
          hipFree(C_cutlass);
          hipFree(B);
          hipFree(A);
          return result;
      }

      // Re-initialize d_ES_b
      result = hipMemcpy(d_ES_b, h_ES_b, nBytes_ES, hipMemcpyHostToDevice);
      if (result != hipSuccess) 
      {
          std::cerr << "Failed to copy h_ES_b matrix to h_ES_b: " << hipGetErrorString(result) << std::endl;
          hipFree(C_reference);
          hipFree(C_cutlass);
          hipFree(B);
          hipFree(A);
          return result;
      }

      // Re-initialize d_ES_c
      hipMemcpy(d_ES_c, h_ES_c, nBytes_ES, hipMemcpyHostToDevice);
      if (result != hipSuccess) {
          std::cerr << "Failed to copy h_ES_c matrix to d_ES_c " << hipGetErrorString(result) << std::endl;
          hipFree(C_reference);
          hipFree(C_cutlass);
          hipFree(B);
          hipFree(A);
          return result;
      }

      // =======================================================================================
      // Launch the kernel to GPU and verify that has not return an error
      // =======================================================================================
      result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, d_ES_a, d_ES_b, d_ES_c);
      hipDeviceSynchronize();
      if (result != hipSuccess) 
      {
          std::cerr << "CUTLASS GEMM kernel failed: " << hipGetErrorString(result) << std::endl;
          hipFree(C_reference);
          hipFree(C_cutlass);
          hipFree(B);
          hipFree(A);
          return result;
      }

      // =======================================================================================
      // Launch the kernel to GPU and verify that has not return an error
      // =======================================================================================
      // Copy to host the values of the ES of A, B and C performed and stored in the GPU device
      result = hipMemcpy(h_ES_a, d_ES_a, nBytes_ES, hipMemcpyDeviceToHost);
      if (result != hipSuccess) {
        std::cerr << "Failed to copy d_ES_a matrix to h_ES_a (return)" << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
      }

      result = hipMemcpy(h_ES_b, d_ES_b, nBytes_ES, hipMemcpyDeviceToHost);
      if (result != hipSuccess) {
        std::cerr << "Failed to copy d_ES_b matrix to h_ES_b (return)" << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
      }

      result = hipMemcpy(h_ES_c, d_ES_c, nBytes_ES, hipMemcpyDeviceToHost);
      if (result != hipSuccess) {
        std::cerr << "Failed to copy d_ES_c matrix to h_ES_c (return)" << hipGetErrorString(result) << std::endl;
        hipFree(C_reference);
        hipFree(C_cutlass);
        hipFree(B);
        hipFree(A);
        return result;
      }

      // =======================================================================================
      // Verify that the ES_a,b,c is the same than the values stored in ES_a_ref,b,c
      // =======================================================================================
      if ((memcmp(h_ES_a,h_ES_a_ref,nBytes_ES)!=0) || (memcmp(h_ES_b,h_ES_b_ref,nBytes_ES)!=0) || (memcmp(h_ES_c,h_ES_c_ref,nBytes_ES)!=0)) {
            ui32_dc_cnt_b += 1u;
      }
      ui32_dc_cnt_all += 1u;

      // =======================================================================================
      // TO flip again the bit that was fliped in host
      // =======================================================================================
      mem_fi(&h_b[0], ui32_idx_bit);
    }

  printf("Number of executions: %d\nDifferents ES: %d\n",ui32_comb_b_max, ui32_dc_cnt_b);
#endif











/* ==============================================================================
    Brief: Store the timing measurements in a file (csv/xlsx)
  ==============================================================================*/
    FILE *p_file;
    char str_file_name[100u];
    char str_file_name_aux[100u] = NAME"_DC.csv";
    snprintf(str_file_name, 100,"%d_%d_%d_",DIM_M,DIM_N,DIM_K);
    strcat(str_file_name,str_file_name_aux);

    char str_file_time[100u];
    time_t time_now = time(NULL);
    struct tm *time_info;
    time_info = localtime(&time_now);
    strftime(str_file_time, sizeof(str_file_time), "%m_%d_%H_%M_%S.csv",time_info);

	if ((p_file = fopen(str_file_name, "a")) == NULL)
	{
		fprintf(stderr, "cannot open file '%s'\n", str_file_name);
		return hipErrorInvalidValue;
	}

	if (!p_file)
	{
		perror("File opening failed");
		return hipErrorInvalidValue;
	}
  
#if (ES_EXP==1)
	fprintf(p_file, "%s,%s,%d,%d,%d,%d,%d,%d,",NAME, str_file_time, DIM_M, DIM_N, DIM_K, ui32_dc_cnt_a,ui32_dc_cnt_all,ui32_comb_a_max);
  // Here i store in the csv file the content of the ES_a_ref,b_ref and c_ref
  for(uint32_t ui32_idx_ES_number=0;ui32_idx_ES_number<3;ui32_idx_ES_number++){
    if(ui32_idx_ES_number==0)       { fprintf(p_file,"\nES_a,");}
    else if (ui32_idx_ES_number==1) { fprintf(p_file,"\nES_b,");}
    else                            { fprintf(p_file,"\nES_c,");}
    
    for(uint32_t ui32_idx_ES=0;ui32_idx_ES<nElem_ES;ui32_idx_ES++){
      if(ui32_idx_ES_number==0)       { fprintf(p_file,"%u,",h_ES_a_ref[ui32_idx_ES]);}
      else if (ui32_idx_ES_number==1) { fprintf(p_file,"%u,",h_ES_b_ref[ui32_idx_ES]);}
      else                            { fprintf(p_file,"%u,",h_ES_c_ref[ui32_idx_ES]);}
    }
  }
  
#else
	fprintf(p_file, "%s,%s,%d,%d,%d,%d,%d,%d,%d,%d,",NAME, str_file_time, DIM_M, DIM_N, DIM_K, ui32_dc_cnt_a,ui32_dc_cnt_b,ui32_dc_cnt_all,ui32_comb_a_max, ui32_comb_b_max);
#endif
  fprintf(p_file,"\n");
  fclose(p_file);

/*
			fprintf(p_file, "%u,", ui32_dc_cnt);
			fprintf(p_file_idx_fi, "\n");
			for (ui32_idx_bit = ui32_idx_bit_aux; ui32_idx_bit < ui32_comb_b_max; ui32_idx_bit++)
			{
				mem_fi(&paf32_mb_fi[0], ui32_idx_bit);
				aui32_dc_value[e_FI_VAR_B] = ptr_fn_smm_technique[e_tech](M, N, K, f32_alpha, (float32_t* const)paf32_ma, (float32_t* const)paf32_mb_fi, (float32_t* const)paf32_mc);
				if (aui32_dc_value[e_FI_VAR_NONE] != aui32_dc_value[e_FI_VAR_B]) {
					ui32_dc_cnt += 1u;
				}
				else {
					fprintf(p_file_idx_fi, "%u,", ui32_idx_bit);
				}
				mem_fi(&paf32_mb_fi[0], ui32_idx_bit);
			}
			fprintf(p_file, "%u,%u,%d,%d,%d,%u,%u,%u,%u", ui32_dc_cnt, (ui32_combinations_a + ui32_combinations_b), M, N, K, ui32_idx_bit_aux, ui32_comb_a_max, ui32_comb_b_max, launch_number);
  */
#endif




 
  /* ==============================================================================
    Brief: Timing measurements. The values are stored in a xlsx file
  ==============================================================================*/
  #if (1==TIMING_EXP)
    uint64_t timing_values[TIME_MEASUREMENT_LOOPS-INITIAL_TIME_MEASUREMENT];

    for (size_t i_loop = 0; i_loop < TIME_MEASUREMENT_LOOPS; i_loop++)
    {
        // Re-initialize ES_a
        result = hipMemcpy(d_ES_a, h_ES_a, nBytes_ES, hipMemcpyHostToDevice);
        if (result != hipSuccess) {
            std::cerr << "Failed to copy h_ES_a matrix to d_ES_a: "
                << hipGetErrorString(result) << std::endl;

            hipFree(C_reference);
            hipFree(C_cutlass);
            hipFree(B);
            hipFree(A);

            return result;
        }

        // Re-initialize ES_b
        result = hipMemcpy(d_ES_b, h_ES_b, nBytes_ES, hipMemcpyHostToDevice);
        if (result != hipSuccess) 
        {
            std::cerr << "Failed to copy h_ES_b matrix to h_ES_b: "
                << hipGetErrorString(result) << std::endl;

            hipFree(C_reference);
            hipFree(C_cutlass);
            hipFree(B);
            hipFree(A);

            return result;
        }

        // Re-initialize ES_c
        hipMemcpy(d_ES_c, h_ES_c, nBytes_ES, hipMemcpyHostToDevice);
        if (result != hipSuccess) {
            std::cerr << "Failed to copy h_ES_c matrix to d_ES_c "
                << hipGetErrorString(result) << std::endl;

            hipFree(C_reference);
            hipFree(C_cutlass);
            hipFree(B);
            hipFree(A);

            return result;
        }

        // Re-initialize C_cutlass
        result = hipMemcpy (C_cutlass,h_c,nBytes_c,hipMemcpyHostToDevice);
        if (result != hipSuccess) {
            std::cerr << "Failed to copy h_c matrix to C_cutlass "
                << hipGetErrorString(result) << std::endl;

            hipFree(C_reference);
            hipFree(C_cutlass);
            hipFree(B);
            hipFree(A);

            return result;
        }

        //GET_TIME(tmr_start);
        clock_gettime(CLOCK_MONOTONIC, &begin);
        result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, d_ES_a, d_ES_b, d_ES_c);
        hipDeviceSynchronize();
        clock_gettime(CLOCK_MONOTONIC, &end);
        
        if (i_loop >= INITIAL_TIME_MEASUREMENT)
        {
            timing_values[i_loop-INITIAL_TIME_MEASUREMENT]= 1e9 * (end.tv_sec - begin.tv_sec) + (end.tv_nsec - begin.tv_nsec);
        }

        // Time measurement (average/standard deviation)

        /*
        if (i_loop > INITIAL_TIME_MEASUREMENT)
		    {
            uint64_t time_ns = 1e9 * (end.tv_sec - begin.tv_sec) + (end.tv_nsec - begin.tv_nsec);
            if(time_ns > time_max_ns)
            {
                time_max_ns = time_ns;
                iteration_max_time = i_loop;
            }
        
        if(i_loop == INITIAL_TIME_MEASUREMENT)
        {
          M_time = time_ns;
          S_time = 0.0;
        }
        double previousM = M_time;
        double previousS = S_time;
        M_time = previousM + (time_ns - previousM) / i_loop;
        S_time = previousS + (time_ns - previousM) * (time_ns - M_time);
		  }
      */

        //GET_TIME(tmr_end);
        //GET_TIME_DIFF(tmr_start, tmr_end, time_interval);
        //time_interval *= ((float64_t)TIME_SEC2USEC);
        //fprintf(p_file, "%f,", time_interval);
        

        if (result != hipSuccess) 
        {
            std::cerr << "CUTLASS GEMM kernel failed: "
                << hipGetErrorString(result) << std::endl;

            hipFree(C_reference);
            hipFree(C_cutlass);
            hipFree(B);
            hipFree(A);

            return result;
        }
    }
/* ==============================================================================
    Brief: Compute the timing values
  ==============================================================================*/
	double time_avg_us = M_time / 1000.0;
	double time_max_us = time_max_ns / 1000.0;
	double variance = S_time / (TIME_MEASUREMENT_LOOPS - INITIAL_TIME_MEASUREMENT);
	double standard_deviation = sqrt(variance) / 1000.0;
	
	//printf("Average time: %.1f us\n", time_avg_us);
	//printf("Maximum time: %.1f us (iteration %lld)\n", time_max_us, iteration_max_time);
	//printf("Standard deviation: %.1f us\n", standard_deviation);
	//printf("%d,%.1f,%.1f,%.1f,%lld\n", threads, time_avg_us, time_max_us, standard_deviation, iteration_max_time);

/* ==============================================================================
    Brief: Store the timing measurements in a file (csv/xlsx)
  ==============================================================================*/
    FILE *p_file;
    char str_file_name[100u];
    char str_file_time[100u];
    char str_file_name_aux[100u] = NAME;
    snprintf(str_file_name, 100,"%d_%d_%d_",DIM_M,DIM_N,DIM_K);
    time_t time_now = time(NULL);

    struct tm *time_info;
    time_info = localtime(&time_now);
    strftime(str_file_time, sizeof(str_file_time), "%m_%d_%H_%M_%S.csv",time_info);
    strcat(str_file_name,str_file_name_aux);


	if ((p_file = fopen(str_file_name, "a")) == NULL)
	{
		fprintf(stderr, "cannot open file '%s'\n", str_file_name);
		return hipErrorInvalidValue;
	}

	if (!p_file)
	{
		perror("File opening failed");
		return hipErrorInvalidValue;
	}
  
	fprintf(p_file, "%s,%s,%d,%d,%d,",NAME, str_file_time, DIM_M, DIM_N, DIM_K);
  /*fprintf(p_file, "%.1f,%.1f,%.1f,%.1f,%lld\n", time_avg_us, time_max_us, standard_deviation,variance, iteration_max_time);
  */
  for (size_t j_loop = 0; j_loop < (TIME_MEASUREMENT_LOOPS-INITIAL_TIME_MEASUREMENT); j_loop++)
  {
      fprintf(p_file, "%lu,", timing_values[j_loop]/1000);
  }
  fprintf(p_file,"\n");
  fclose(p_file);


  result = hipMemcpy (C_reference,h_c,nBytes_c,hipMemcpyHostToDevice);
 if (result != hipSuccess) {
    std::cerr << "Failed to copy h_c matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // =================================================================================================
  //                            Launch CUTLASS GEMM.
  // =================================================================================================
  // Copy to host the values of the ES of A, B and C performed and stored in the GPU device
  result = hipMemcpy(h_ES_a, d_ES_a, nBytes_ES, hipMemcpyDeviceToHost);
  result = hipMemcpy(h_ES_b, d_ES_b, nBytes_ES, hipMemcpyDeviceToHost);
  result = hipMemcpy(h_ES_c, d_ES_c, nBytes_ES, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel 2 failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

   d_ES.A = 0;
   d_ES.B = 0;
   d_ES.C = 0;

/* To use with XOR
  for(int i=0;i<nElem_ES;i++){
    printf("ES_a[%i] = %u \t ES_b = %u \t ES_c = %u\n",i,h_ES_a[i],h_ES_b[i],h_ES_c[i]);
    d_ES.A ^= h_ES_a[i];
    d_ES.B ^= h_ES_b[i];
    d_ES.C ^= h_ES_c[i];
  }
printf("Final ES (GPU)\n Es_a =%12u \t Es_b =%12u \t Es_c =%12u \n", d_ES.A, d_ES.B, d_ES.C);
*/

/* To use with One's complement checksum
	// One's complement checksum 
	ui64_to_ui32_t Ones_Checksum_a,
		Ones_Checksum_b,
		Ones_Checksum_c,
		Ones_Checksum;

	Ones_Checksum_a.ui64 = 0u;
	Ones_Checksum_b.ui64 = 0u;
	Ones_Checksum_c.ui64 = 0u;
  
  printf("Initial values a=%u, b=%u, c=%u\n",Ones_Checksum_a.ui32[0],Ones_Checksum_b.ui32[0], Ones_Checksum_c.ui32[0]);
  for(int i=0;i<nElem_ES;i++){
    Ones_Checksum_a.ui64 += (uint64_t) h_ES_a[i];
		Ones_Checksum_a.ui32[0] += Ones_Checksum_a.ui32[1];
		Ones_Checksum_a.ui32[0] = ~Ones_Checksum_a.ui32[0];
    Ones_Checksum_a.ui32[1] = 0;
    Ones_Checksum_b.ui64 += (uint64_t) h_ES_b[i];
		Ones_Checksum_b.ui32[0] += Ones_Checksum_b.ui32[1];
		Ones_Checksum_b.ui32[0] = ~Ones_Checksum_b.ui32[0];
    Ones_Checksum_b.ui32[1] = 0;
    Ones_Checksum_c.ui64 += (uint64_t) h_ES_c[i];
		Ones_Checksum_c.ui32[0] += Ones_Checksum_c.ui32[1];
		Ones_Checksum_c.ui32[0] = ~Ones_Checksum_c.ui32[0];
    Ones_Checksum_c.ui32[1] = 0;
    //printf("ES_a[%i] = %u \t ES_b = %u \t ES_c = %u \n",i,h_ES_a[i],h_ES_b[i],h_ES_c[i]);
  }
    d_ES.A = Ones_Checksum_a.ui32[0];
    d_ES.B = Ones_Checksum_b.ui32[0];
    d_ES.C = Ones_Checksum_c.ui32[0];
printf("Final ES (GPU)\n Es_a =%12u \t Es_b =%12u \t Es_c =%12u \n", d_ES.A, d_ES.B, d_ES.C);
*/

/* To use with Fletcher



// To use with CRC
  for(int uint32_i=0;uint32_i<nElem_ES;uint32_i++){
     //printf("ES_a[%i] = %u \t ES_b = %u \t ES_c = %u \n",i,h_ES_a[i],h_ES_b[i],h_ES_c[i]);
    d_ES.A = singletable_crc32c_ui32(d_ES.A, h_ES_a[uint32_i]);
    d_ES.B = singletable_crc32c_ui32(d_ES.B, h_ES_b[uint32_i]);
    d_ES.C = singletable_crc32c_ui32(d_ES.C, h_ES_c[uint32_i]);
  }
printf("Final ES (GPU)\n Es_a =%12u \t Es_b =%12u \t Es_c =%12u \n", d_ES.A, d_ES.B, d_ES.C);
*/

// Verify that with a sequential implementation we obtain the same value
//h_ES = smm_xor_internal((uint32_t) M,(uint32_t) N,(uint32_t) K, (float32_t) 1.0f, h_a, h_b, h_c);
printf(" Value of &B[0]=%p\n", d_ES_a);
h_ES = smm_ones_internal((uint32_t) M,(uint32_t) N,(uint32_t) K, (float32_t) 1.0f, h_a, h_b, h_c);

//printf("Final ES_b(CPU)\n Es_a =%u \t Es_b =%u \t Es_c =%u \n", h_ES.A, h_ES.B, h_ES.C);

#endif

  //
  // Verify.
  //

  // Launch reference GEMM
  result = ReferenceGemm(M, N, K, alpha, A, lda, B, ldb, beta, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<float> host_cutlass(ldc * N, 0);
  std::vector<float> host_reference(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);
   result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }




  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C_cutlass);
  hipFree(B);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  }

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_gemm example.
//
// usage:
//
//   00_basic_gemm <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

  //
  // Parse the command line to obtain GEMM dimensions and scalar values.
  //

  // GEMM problem dimensions.
  int problem[3] = { DIM_M, DIM_N, DIM_K  };

  for (int i = 1; i < argc && i < 4; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  float scalars[2] = { 1, 0 };

  for (int i = 4; i < argc && i < 6; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 4];
  }

  //
  // Run the CUTLASS GEMM test.
  //

  hipError_t result = TestCutlassGemm (
    problem[0],     // GEMM M dimension
    problem[1],     // GEMM N dimension
    problem[2],     // GEMM K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }
  // Reset Device
  hipDeviceReset();

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
