#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS GEMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This is kernel computes
  the general matrix product (GEMM) using single-precision floating-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 128x128x8 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SGEMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

  This example has delibrately been kept similar to the basic_gemm example from cutass-1.3 to 
  highlight the minimum amount of differences needed to transition to cutlass-2.0.

  Cutlass-1.3 sgemm: https://github.com/NVIDIA/cutlass/blob/master/examples/00_basic_gemm/basic_gemm.cu
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
// Helper methods to check for errors
#include "helper.h"



#define PUT_IN_REGISTER								 /* dummy definition  for Windows 32 */

typedef float    float32_t;
typedef union ui64_to_ui32 {
	uint64_t ui64;
	uint32_t ui32[2];
} ui64_to_ui32_t;

typedef float    float32_t;
typedef double   float64_t;
typedef void     void_t;

static void_t matrix2rand(float32_t * paf32_matrix, uint32_t ui32_max_rows, uint32_t ui32_max_columns)
{
	uint32_t ui32_idx;

	for (ui32_idx = 0u; ui32_idx < (ui32_max_rows * ui32_max_columns); ui32_idx++)
	{
		*paf32_matrix++ = (float32_t)rand();
	}
}

//
// CUTLASS includes needed for single-precision GEMM kernel
//

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/gemm.h"

// Include Smmm.h (MMM employed in the previous paper)
//#include "cutlass/sequential_MMM/Smmm.h"

  // Definition of an struct to store th values of the Execution Signatures
  struct ESs{
    uint32_t A;
    uint32_t B;
    uint32_t C;
  };

 ESs smm_xor_internal(uint32_t ui32_m, uint32_t ui32_n, uint32_t ui32_k, float32_t f32_alpha,  float32_t*  paf32_ma,  float32_t*  paf32_mb, float32_t *paf32_mc)
{
	uint32_t ui32_idx_i = 0u,
		ui32_idx_j = 0u,
		ui32_idx_k = 0u,
		ui32_idx_a = 0u,
		ui32_idx_b = 0u,
		ui32_idx_c = 0u,
		ui32_idx_b_ref = 0u,
		ui32_idx_c_ref = 0u;

	float32_t f32_a_part = 0.0f,
		f32_b = 0.0f,
		f32_c = 0.0f;

	/* XOR checksum */
  struct ESs ES;
  ES.A = 0u;
  ES.B = 0u;
  ES.C = 0u;


	// Verification of the input values
	assert(paf32_ma != NULL);
	assert(paf32_mb != NULL);
	assert(paf32_mc != NULL);

	for (ui32_idx_i = 0u; ui32_idx_i < ui32_m; ui32_idx_i++)
	{
		ui32_idx_b_ref = 0u;
		for (ui32_idx_k = 0u; ui32_idx_k < ui32_k; ui32_idx_k++, ui32_idx_a++)
		{
			PUT_IN_REGISTER f32_a_part = f32_alpha * paf32_ma[ui32_idx_a];
			ES.A ^= (uint32_t) *((uint32_t*)&f32_a_part);

			for (ui32_idx_j = 0u, ui32_idx_b = ui32_idx_b_ref, ui32_idx_c = ui32_idx_c_ref; ui32_idx_j < ui32_n; ui32_idx_j++, ui32_idx_b++, ui32_idx_c++)
			{
				f32_b = paf32_mb[ui32_idx_b];
				paf32_mc[ui32_idx_c] += f32_a_part * f32_b;
				f32_c = paf32_mc[ui32_idx_c];

				/* XOR checksum */
				ES.B ^= (uint32_t) *((uint32_t*)&f32_b);
        //printf("Value[%u]=%u\n",ui32_idx_b,(uint32_t) *((uint32_t*)&f32_b));
				ES.C ^= (uint32_t) *((uint32_t*)&f32_c);
			}
      //printf("%u\n",ES.B);
			ui32_idx_b_ref += ui32_n;
		}
		ui32_idx_c_ref += ui32_n;
	}
	//ui32_xor = (ui32_xor_a ^ ui32_xor_b) ^ ui32_xor_c;
	return ES;
}


/*==============================================================================================================
**									Name: smm_ones_internal
==============================================================================================================*/
/*!
** @brief Matrix-matrix multiplication (MMM) with One's checksum in the internal loop
**
** @param[in] ui32_m 		Number of matrix A rows 								[0…ui32_m]
** @param[in] ui32_n 		Number of matrix B columns 								[0…ui32_n]
** @param[in] ui32_k 		Number of matrix A columns / Number of matrix B rows 	[0…ui32_k]
** @param[in] f32_alpha Correction factor
** @param[in] paf32_ma 	Pointer to the first position of an array of floats (A matrix direction)
** @param[in] paf32_mb 	Pointer to the first position of an array of floats (B matrix direction)
** @param[in] paf32_mc 	Pointer to the first position of an array of floats (B matrix direction)
**
** @return uint32_t  	ui32_xor	Return the Execution signature of the MMM
==============================================================================================================*/
ESs smm_ones_internal(uint32_t ui32_m, uint32_t ui32_n, uint32_t ui32_k, float32_t f32_alpha,  float32_t*  paf32_ma,  float32_t*  paf32_mb, float32_t*  paf32_mc)
{
	uint32_t ui32_idx_i = 0u,
		ui32_idx_j = 0u,
		ui32_idx_k = 0u,
		ui32_idx_a = 0u,
		ui32_idx_b = 0u,
		ui32_idx_c = 0u,
		ui32_idx_b_ref = 0u,
		ui32_idx_c_ref = 0u;

	float32_t f32_a_part = 0.0f,
		f32_b = 0.0f,
		f32_c = 0.0f;

	/* One's complement checksum */
	ui64_to_ui32_t Ones_Checksum_a,
		Ones_Checksum_b,
		Ones_Checksum_c,
		Ones_Checksum;

	Ones_Checksum_a.ui64 = 0u;
	Ones_Checksum_b.ui64 = 0u;
	Ones_Checksum_c.ui64 = 0u;


	// Verification of the input values
	assert(paf32_ma != NULL);
	assert(paf32_mb != NULL);
	assert(paf32_mc != NULL);

	for (ui32_idx_i = 0u; ui32_idx_i < ui32_m; ui32_idx_i++)
	{
		ui32_idx_b_ref = 0u;
		for (ui32_idx_k = 0u; ui32_idx_k < ui32_k; ui32_idx_k++, ui32_idx_a++)
		{
			PUT_IN_REGISTER f32_a_part = f32_alpha * paf32_ma[ui32_idx_a];
			Ones_Checksum_a.ui64 += (uint64_t) * ((uint32_t*)&f32_a_part);
			Ones_Checksum_a.ui32[0] += Ones_Checksum_a.ui32[1];
			Ones_Checksum_a.ui32[0] = ~Ones_Checksum_a.ui32[0];
      Ones_Checksum_a.ui32[1] = 0;

			for (ui32_idx_j = 0u, ui32_idx_b = ui32_idx_b_ref, ui32_idx_c = ui32_idx_c_ref; ui32_idx_j < ui32_n;
				ui32_idx_j++, ui32_idx_b++, ui32_idx_c++)
			{
				f32_b = paf32_mb[ui32_idx_b];
				paf32_mc[ui32_idx_c] += f32_a_part * f32_b;
				f32_c = paf32_mc[ui32_idx_c];
				/* One's complement checksum */
				Ones_Checksum_b.ui64 += (uint64_t) * ((uint32_t*)&f32_b);
				Ones_Checksum_b.ui32[0] += Ones_Checksum_b.ui32[1];
				Ones_Checksum_b.ui32[0] = ~Ones_Checksum_b.ui32[0];

				Ones_Checksum_c.ui64 += (uint64_t) * ((uint32_t*)&f32_c);
				Ones_Checksum_c.ui32[0] += Ones_Checksum_c.ui32[1];
				Ones_Checksum_c.ui32[0] = ~Ones_Checksum_c.ui32[0];

        Ones_Checksum_b.ui32[1] = 0;
        Ones_Checksum_c.ui32[1] = 0;
			}
			ui32_idx_b_ref += ui32_n;
		}
		ui32_idx_c_ref += ui32_n;
	}
  /*
	Ones_Checksum.ui64 = (Ones_Checksum_a.ui64 + Ones_Checksum_b.ui64);
	Ones_Checksum.ui32[0] += Ones_Checksum.ui32[1];
	Ones_Checksum.ui32[0] = ~Ones_Checksum.ui32[0];
	Ones_Checksum.ui64 += Ones_Checksum_c.ui64;
	Ones_Checksum.ui32[0] += Ones_Checksum.ui32[1];
	Ones_Checksum.ui32[0] = ~Ones_Checksum.ui32[0];
  */
   struct ESs ES;
  ES.A = Ones_Checksum_a.ui32[0];
  ES.B = Ones_Checksum_b.ui32[0];
  ES.C = Ones_Checksum_c.ui32[0];

	return ES;
}


///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc,
  uint32_t *d_ES_a,
  uint32_t *d_ES_b,
  uint32_t *d_ES_c) {

  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h`

  using ColumnMajor = cutlass::layout::ColumnMajor;

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                  ColumnMajor,  // Layout of A matrix
                                                  float,        // Data-type of B matrix
                                                  ColumnMajor,  // Layout of B matrix
                                                  float,        // Data-type of C matrix
                                                  ColumnMajor>; // Layout of C matrix

  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  //printf("\n Direction of h_ES_0: %p and value: %f \n", (void *) h_ES_0, h_ES_0[4]);

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta},// Scalars used in the Epilogue
                              d_ES_a,   // Pointer to d_ES_a
                              d_ES_b,   // Pointer to d_ES_b
                              d_ES_c);  // Pointer to d_ES_c

  // Code included by JFdez: I have to include in args variable this: d_ES_0 and d_ES_1

  //
  // Launch the CUTLASS GEMM kernel.
  //
  
  cutlass::Status status = gemm_operator(args);

  //
  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  float *matrix,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * rows;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);
    /*if(((offset+1)%columns)==0){
      printf("\n");
    }
    printf("Matrix[%d]=%f \t",offset,value);
    */
    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(float *matrix, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, rows, columns, seed);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(float **matrix, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(float) * rows * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  // cHANGED BY Jfdez
  result = InitializeMatrix(*matrix, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Naive reference GEMM computation.
__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    float accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

/// Reference GEMM computation.
hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to GEMM kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(float) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  float *A;
  float *B;
  float *C_cutlass;
  float *C_reference;

  // =============================================================
  // Author:  Javier Fdez
  // Date:    2021/08/17
  // Summary: In the following chunk of code are initialized in 
  //          CPU and GPU the variables ES_0 and ES_1 in which are 
  //          stored the Execution Signatures (1 per thread*SMP)
  // =============================================================

  // Definition of an struct with the values of the ES:
  struct ESs h_ES;
  struct ESs d_ES;

  // Define the number of elements of the ES 
  uint32_t nElem_ES = 256;
  size_t nBytes_ES = nElem_ES * sizeof(uint32_t);

  // Define pointers to ES_a, ES_b and ES_c in CPU (host)
  uint32_t *h_ES_a;
  uint32_t *h_ES_b;
  uint32_t *h_ES_c;

  // Allocate ES_0 y ES_1 in CPU 
  h_ES_a = (uint32_t *) malloc(nBytes_ES);
  h_ES_b = (uint32_t *) malloc(nBytes_ES);
  h_ES_c = (uint32_t *) malloc(nBytes_ES);

  // Initialice to 0 all values of ES_a, ES_b and ES_c
  memset(h_ES_a,0,nBytes_ES);
  memset(h_ES_b,0,nBytes_ES);
  memset(h_ES_c,0,nBytes_ES);

/*
h_ES_0[1] = 1;
h_ES_0[2] = 2;


for(int i=0;i<nElem_ES;i++){
  printf("ES[%i] = %u \n",i,h_ES_a[i]);
}
*/


  // Define pointers to ES_a, ES_b and ES_c in GPU (device)
  uint32_t *d_ES_a;
  uint32_t *d_ES_b;
  uint32_t *d_ES_c;

  // Allocate ES_0 y ES_1 in GPU
  hipMalloc((uint32_t **) &d_ES_a, nBytes_ES);
  hipMalloc((uint32_t **) &d_ES_b, nBytes_ES);
  hipMalloc((uint32_t **) &d_ES_c, nBytes_ES);

  // Transfer data from host to device (first time it has no sense, it could
  // be directly initilized in GPU, but it will not be always initially zero)
  hipMemcpy(d_ES_a, h_ES_a, nBytes_ES, hipMemcpyHostToDevice);
  hipMemcpy(d_ES_b, h_ES_b, nBytes_ES, hipMemcpyHostToDevice);
  hipMemcpy(d_ES_c, h_ES_c, nBytes_ES, hipMemcpyHostToDevice);

  /* ==============================================================================
  Brief: Initialization of A, B, C_reference and C_cutlass. 
  Hypothesis: the values of A and B follow a pattern in the initialization if it 
  is done in with the GPU. This sequential implementation aid to avoid this.
  ==============================================================================*/
  uint32_t nBytes_a = M * K * sizeof(float);
  uint32_t nBytes_b = K * N * sizeof(float);
  uint32_t nBytes_c = M * N * sizeof(float);

  // Define pointers to h_a, h_b and h_c in CPU (host)
  float *h_a;
  float *h_b;
  float *h_c;

  // Allocate h_a, h_b and h_c in CPU 
  h_a     = (float *) malloc(nBytes_a);
  h_b     = (float *) malloc(nBytes_b);
  h_c     = (float *) malloc(nBytes_c);

  // Allocate memory on GPU
  result =  hipMalloc(&A, nBytes_a);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate A: "
      << hipGetErrorString(result) << std::endl;}

  result =  hipMalloc(&B, nBytes_b);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate B: "
      << hipGetErrorString(result) << std::endl;}

  result =  hipMalloc(&C_cutlass, nBytes_c);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate C_cutlass: "
      << hipGetErrorString(result) << std::endl;}

  result =  hipMalloc(&C_reference, nBytes_c);
  if (result != hipSuccess) {
    std::cerr << "Failed to allocate C_reference: "
      << hipGetErrorString(result) << std::endl;}
  

  // Initialice to 0 all values of h_c and h_c_reference
  memset(h_c,0,nBytes_c);
  
  // Initialization of the values of h_a and h_b
  matrix2rand(h_a,M,K);
  matrix2rand(h_b,K,N);

  // Copy the values initialized and stored in host to the device (h_a -> A, h_b -> B ...)
  result = hipMemcpy (A,h_a,nBytes_a,hipMemcpyHostToDevice);
 if (result != hipSuccess) {
    std::cerr << "Failed to copy h_a matrix to A: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy (B,h_b,nBytes_b,hipMemcpyHostToDevice);
 if (result != hipSuccess) {
    std::cerr << "Failed to copy h_b matrix to B: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy (C_reference,h_c,nBytes_c,hipMemcpyHostToDevice);
 if (result != hipSuccess) {
    std::cerr << "Failed to copy h_c matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy (C_cutlass,h_c,nBytes_c,hipMemcpyHostToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }




/*
  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  //printf("Matrix A:\n");
  result = AllocateMatrix(&A, M, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }
  //printf("Matrix B:\n");
  result = AllocateMatrix(&B, K, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_cutlass, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  result = AllocateMatrix(&C_reference, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    hipFree(C_cutlass);
    return result;
  }

  // Define pointers to matrices in CPU device memory
  float *h_A;
  float *h_B;

  // Allocate h_A and h_B in CPU 
  h_A = (float *) malloc(M*K*sizeof(float));
  h_B = (float *) malloc(N*K*sizeof(float));

  // Copy from device to host the value of the matrices A and B
  result = hipMemcpy(h_A, A, M*K*sizeof(float), hipMemcpyDeviceToHost);
  result = hipMemcpy(h_B, B,  N*K*sizeof(float), hipMemcpyDeviceToHost);

  // Define pointers to C_reference_sequential
  float *C_reference_sequential;

  // Allocate ES_0 y ES_1 in CPU 
  C_reference_sequential = (float *) malloc(M*N* sizeof(float32_t));

  // Initialice to 0 all values of ES_0 and ES_1
  memset(C_reference_sequential,0,M*N* sizeof(float32_t));


  result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }
*/


  //
  // Launch CUTLASS GEMM.
  //

  //result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, d_ES_a, d_ES_b, d_ES_c);


  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host the values of the ES of A, B and C performed and stored in the GPU device
  result = hipMemcpy(h_ES_a, d_ES_a, nBytes_ES, hipMemcpyDeviceToHost);
  result = hipMemcpy(h_ES_b, d_ES_b, nBytes_ES, hipMemcpyDeviceToHost);
  result = hipMemcpy(h_ES_c, d_ES_c, nBytes_ES, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel 2 failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

   d_ES.A = 0;
   d_ES.B = 0;
   d_ES.C = 0;

/* To use with XOR
  for(int i=0;i<nElem_ES;i++){
    printf("ES_a[%i] = %u \t ES_b = %u \t ES_c = %u\n",i,h_ES_a[i],h_ES_b[i],h_ES_c[i]);
    d_ES.A ^= h_ES_a[i];
    d_ES.B ^= h_ES_b[i];
    d_ES.C ^= h_ES_c[i];
  }
printf("Final ES (GPU)\n Es_a =%12u \t Es_b =%12u \t Es_c =%12u \n", d_ES.A, d_ES.B, d_ES.C);
*/

	/* One's complement checksum */
	ui64_to_ui32_t Ones_Checksum_a,
		Ones_Checksum_b,
		Ones_Checksum_c,
		Ones_Checksum;

	Ones_Checksum_a.ui64 = 0u;
	Ones_Checksum_b.ui64 = 0u;
	Ones_Checksum_c.ui64 = 0u;
  
  printf("Initial values a=%u, b=%u, c=%u\n",Ones_Checksum_a.ui32[0],Ones_Checksum_b.ui32[0], Ones_Checksum_c.ui32[0]);
  for(int i=0;i<nElem_ES;i++){
    Ones_Checksum_a.ui64 += (uint64_t) h_ES_a[i];
		Ones_Checksum_a.ui32[0] += Ones_Checksum_a.ui32[1];
		Ones_Checksum_a.ui32[0] = ~Ones_Checksum_a.ui32[0];
    Ones_Checksum_a.ui32[1] = 0;
    Ones_Checksum_b.ui64 += (uint64_t) h_ES_b[i];
		Ones_Checksum_b.ui32[0] += Ones_Checksum_b.ui32[1];
		Ones_Checksum_b.ui32[0] = ~Ones_Checksum_b.ui32[0];
    Ones_Checksum_b.ui32[1] = 0;
    Ones_Checksum_c.ui64 += (uint64_t) h_ES_c[i];
		Ones_Checksum_c.ui32[0] += Ones_Checksum_c.ui32[1];
		Ones_Checksum_c.ui32[0] = ~Ones_Checksum_c.ui32[0];
    Ones_Checksum_c.ui32[1] = 0;
    printf("ES_a[%i] = %u \t ES_b = %u \t ES_c = %u \n",i,h_ES_a[i],h_ES_b[i],h_ES_c[i]);
  }
    d_ES.A = Ones_Checksum_a.ui32[0];
    d_ES.B = Ones_Checksum_b.ui32[0];
    d_ES.C = Ones_Checksum_c.ui32[0];
printf("Final ES (GPU)\n Es_a =%12u \t Es_b =%12u \t Es_c =%12u \n", d_ES.A, d_ES.B, d_ES.C);


// Verify that with a sequential implementation we obtain the same value
//h_ES = smm_xor_internal((uint32_t) M,(uint32_t) N,(uint32_t) K, (float32_t) 1.0f, h_a, h_b, h_c);
printf(" Value of &B[0]=%p\n", d_ES_a);
h_ES = smm_ones_internal((uint32_t) M,(uint32_t) N,(uint32_t) K, (float32_t) 1.0f, h_a, h_b, h_c);

printf("Final ES_b(CPU)\n Es_a =%u \t Es_b =%u \t Es_c =%u \n", h_ES.A, h_ES.B, h_ES.C);



  //
  // Verify.
  //

  // Launch reference GEMM
  result = ReferenceGemm(M, N, K, alpha, A, lda, B, ldb, beta, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<float> host_cutlass(ldc * N, 0);
  std::vector<float> host_reference(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);
   result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C_cutlass);
  hipFree(B);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  }

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_gemm example.
//
// usage:
//
//   00_basic_gemm <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

  //F
  // Parse the command line to obtain GEMM dimensions and scalar values.
  //

  // GEMM problem dimensions.
  int problem[3] = { 128, 128, 128  };

  for (int i = 1; i < argc && i < 4; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  float scalars[2] = { 1, 0 };

  for (int i = 4; i < argc && i < 6; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 4];
  }

  //
  // Run the CUTLASS GEMM test.
  //

  hipError_t result = TestCutlassGemm(
    problem[0],     // GEMM M dimension
    problem[1],     // GEMM N dimension
    problem[2],     // GEMM K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
