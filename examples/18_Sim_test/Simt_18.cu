#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS GEMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Gemm template is instantiated in the function CutlassSgemmNN. This is kernel computes
  the general matrix product (GEMM) using single-precision floating-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 128x128x8 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SGEMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

  This example has delibrately been kept similar to the basic_gemm example from cutass-1.3 to 
  highlight the minimum amount of differences needed to transition to cutlass-2.0.

  Cutlass-1.3 sgemm: https://github.com/NVIDIA/cutlass/blob/master/examples/00_basic_gemm/basic_gemm.cu
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
typedef float    float32_t;
#define PUT_IN_REGISTER								 /* dummy definition  for Windows 32 */
// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for single-precision GEMM kernel
//

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/gemm.h"

// Include Smmm.h (MMM employed in the previous paper)
//#include "cutlass/sequential_MMM/Smmm.h"

  // Definition of an struct to store th values of the Execution Signatures
  struct ESs{
    uint32_t A;
    uint32_t B;
    uint32_t C;
  };

 ESs smm_xor_internal(uint32_t ui32_m, uint32_t ui32_n, uint32_t ui32_k, float32_t f32_alpha,  float32_t*  paf32_ma,  float32_t*  paf32_mb, float32_t *paf32_mc)
{
	uint32_t ui32_idx_i = 0u,
		ui32_idx_j = 0u,
		ui32_idx_k = 0u,
		ui32_idx_a = 0u,
		ui32_idx_b = 0u,
		ui32_idx_c = 0u,
		ui32_idx_b_ref = 0u,
		ui32_idx_c_ref = 0u;

	float32_t f32_a_part = 0.0f,
		f32_b = 0.0f,
		f32_c = 0.0f;

	/* XOR checksum */
  struct ESs ES;
  ES.A = 0u;
  ES.B = 0u;
  ES.C = 0u;


	// Verification of the input values
	assert(paf32_ma != NULL);
	assert(paf32_mb != NULL);
	assert(paf32_mc != NULL);

	for (ui32_idx_i = 0u; ui32_idx_i < ui32_m; ui32_idx_i++)
	{
		ui32_idx_b_ref = 0u;
		for (ui32_idx_k = 0u; ui32_idx_k < ui32_k; ui32_idx_k++, ui32_idx_a++)
		{
			PUT_IN_REGISTER f32_a_part = f32_alpha * paf32_ma[ui32_idx_a];
			ES.A ^= (uint32_t) *((uint32_t*)&f32_a_part);

			for (ui32_idx_j = 0u, ui32_idx_b = ui32_idx_b_ref, ui32_idx_c = ui32_idx_c_ref; ui32_idx_j < ui32_n; ui32_idx_j++, ui32_idx_b++, ui32_idx_c++)
			{
				f32_b = paf32_mb[ui32_idx_b];
				paf32_mc[ui32_idx_c] += f32_a_part * f32_b;
				f32_c = paf32_mc[ui32_idx_c];

				/* XOR checksum */
				ES.B ^= (uint32_t) *((uint32_t*)&f32_b);
        //printf("Value[%u]=%u\n",ui32_idx_b,(uint32_t) *((uint32_t*)&f32_b));
				ES.C ^= (uint32_t) *((uint32_t*)&f32_c);
			}
      //printf("%u\n",ES.B);
			ui32_idx_b_ref += ui32_n;
		}
		ui32_idx_c_ref += ui32_n;
	}
	//ui32_xor = (ui32_xor_a ^ ui32_xor_b) ^ ui32_xor_c;
	return ES;
}



///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc,
  uint32_t *d_ES_a,
  uint32_t *d_ES_b,
  uint32_t *d_ES_c) {

  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h`

  using ColumnMajor = cutlass::layout::ColumnMajor;

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                                  ColumnMajor,  // Layout of A matrix
                                                  float,        // Data-type of B matrix
                                                  ColumnMajor,  // Layout of B matrix
                                                  float,        // Data-type of C matrix
                                                  ColumnMajor>; // Layout of C matrix

  // Define a CUTLASS GEMM type
  CutlassGemm gemm_operator;

  //printf("\n Direction of h_ES_0: %p and value: %f \n", (void *) h_ES_0, h_ES_0[4]);

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source matrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta},// Scalars used in the Epilogue
                              d_ES_a,   // Pointer to d_ES_a
                              d_ES_b,   // Pointer to d_ES_b
                              d_ES_c);  // Pointer to d_ES_c

  // Code included by JFdez: I have to include in args variable this: d_ES_0 and d_ES_1

  //
  // Launch the CUTLASS GEMM kernel.
  //
  
  cutlass::Status status = gemm_operator(args);

  //
  // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  float *matrix,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * rows;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    float value = float(((offset + seed) * k % m) - m / 2);
    /*if(((offset+1)%columns)==0){
      printf("\n");
    }
    printf("Matrix[%d]=%f \t",offset,value);*/
    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(float *matrix, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, rows, columns, seed);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(float **matrix, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(float) * rows * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  // cHANGED BY Jfdez
  result = InitializeMatrix(*matrix, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Naive reference GEMM computation.
__global__ void ReferenceGemm_kernel(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    float accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

/// Reference GEMM computation.
hipError_t ReferenceGemm(
  int M,
  int N,
  int K,
  float alpha,
  float const *A,
  int lda,
  float const *B,
  int ldb,
  float beta,
  float *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceGemm_kernel<<< grid, block >>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, float alpha, float beta) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to GEMM kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(float) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  float *A;
  float *B;
  float *C_cutlass;
  float *C_reference;

  // =============================================================
  // Author:  Javier Fdez
  // Date:    2021/08/17
  // Summary: In the following chunk of code are initialized in 
  //          CPU and GPU the variables ES_0 and ES_1 in which are 
  //          stored the Execution Signatures (1 per thread*SMP)
  // =============================================================

  // Definition of an struct with the values of the ES:
  struct ESs h_ES;
  struct ESs d_ES;

  // Define the number of elements of the ES 
  uint32_t nElem_ES = 32;
  size_t nBytes_ES = nElem_ES * sizeof(uint32_t);

  // Define pointers to ES_a, ES_b and ES_c in CPU (host)
  uint32_t *h_ES_a;
  uint32_t *h_ES_b;
  uint32_t *h_ES_c;

  // Allocate ES_0 y ES_1 in CPU 
  h_ES_a = (uint32_t *) malloc(nBytes_ES);
  h_ES_b = (uint32_t *) malloc(nBytes_ES);
  h_ES_c = (uint32_t *) malloc(nBytes_ES);

  // Initialice to 0 all values of ES_a, ES_b and ES_c
  memset(h_ES_a,0,nBytes_ES);
  memset(h_ES_b,0,nBytes_ES);
  memset(h_ES_c,0,nBytes_ES);

/*
h_ES_0[1] = 1;
h_ES_0[2] = 2;


for(int i=0;i<nElem_ES;i++){
  printf("ES[%i] = %u \n",i,h_ES_a[i]);
}
*/


  // Define pointers to ES_a, ES_b and ES_c in GPU (device)
  uint32_t *d_ES_a;
  uint32_t *d_ES_b;
  uint32_t *d_ES_c;

  // Allocate ES_0 y ES_1 in GPU
  hipMalloc((uint32_t **) &d_ES_a, nBytes_ES);
  hipMalloc((uint32_t **) &d_ES_b, nBytes_ES);
  hipMalloc((uint32_t **) &d_ES_c, nBytes_ES);

  // Transfer data from host to device (first time it has no sense, it could
  // be directly initilized in GPU, but it will not be always initially zero)
  hipMemcpy(d_ES_a, h_ES_a, nBytes_ES, hipMemcpyHostToDevice);
  hipMemcpy(d_ES_b, h_ES_b, nBytes_ES, hipMemcpyHostToDevice);
  hipMemcpy(d_ES_c, h_ES_c, nBytes_ES, hipMemcpyHostToDevice);

  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  //printf("Matrix A:\n");
  result = AllocateMatrix(&A, M, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }
  //printf("Matrix B:\n");
  result = AllocateMatrix(&B, K, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_cutlass, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  result = AllocateMatrix(&C_reference, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    hipFree(C_cutlass);
    return result;
  }

  // Define pointers to matrices in CPU device memory
  float *h_A;
  float *h_B;

  // Allocate h_A and h_B in CPU 
  h_A = (float *) malloc( M*K*sizeof(float));
  h_B = (float *) malloc( N*K*sizeof(float));

  // Copy from device to host the value of the matrices A and B
  result = hipMemcpy(h_A, A, M*K*sizeof(float), hipMemcpyDeviceToHost);
  result = hipMemcpy(h_B, B,  N*K*sizeof(float), hipMemcpyDeviceToHost);

  // Define pointers to C_reference_sequential
  float *C_reference_sequential;

  // Allocate ES_0 y ES_1 in CPU 
  C_reference_sequential = (float *) malloc(M*N* sizeof(float32_t));

  // Initialice to 0 all values of ES_0 and ES_1
  memset(C_reference_sequential,0,M*N* sizeof(float32_t));


  result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Launch CUTLASS GEMM.
  //

  //result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc);
  result = CutlassSgemmNN(M, N, K, alpha, A, lda, B, ldb, beta, C_cutlass, ldc, d_ES_a, d_ES_b, d_ES_c);

  // Copy to host the values of the ES of A, B and C performed and stored in the GPU device
  result = hipMemcpy(h_ES_a, d_ES_a, nBytes_ES, hipMemcpyDeviceToHost);
  result = hipMemcpy(h_ES_b, d_ES_b, nBytes_ES, hipMemcpyDeviceToHost);
  result = hipMemcpy(h_ES_c, d_ES_c, nBytes_ES, hipMemcpyDeviceToHost);

   d_ES.A = 0;
   d_ES.B = 0;
   d_ES.C = 0;

  for(int i=0;i<nElem_ES;i++){
    printf("ES_a[%i] = %u \n",i,h_ES_a[i]);
    d_ES.A ^= h_ES_a[i];
    d_ES.B ^= h_ES_b[i];
    d_ES.C ^= h_ES_c[i];
  }
printf("Final ES_b(GPU)\n Es_a =%u \t Es_b =%u \t Es_c =%u \n", d_ES.A, d_ES.B, d_ES.C);
 
// Verify that with a sequential implementation we obtain the same value
h_ES = smm_xor_internal((uint32_t) M,(uint32_t) N,(uint32_t) K, (float32_t) 1.0f, h_A, h_B, C_reference_sequential);
printf("Final ES_b(CPU)\n Es_a =%u \t Es_b =%u \t Es_c =%u \n", h_ES.A, h_ES.B, h_ES.C);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Verify.
  //

  // Launch reference GEMM
  result = ReferenceGemm(M, N, K, alpha, A, lda, B, ldb, beta, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference GEMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<float> host_cutlass(ldc * N, 0);
  std::vector<float> host_reference(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);
   result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference GEMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C_cutlass);
  hipFree(B);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  }

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_gemm example.
//
// usage:
//
//   00_basic_gemm <M> <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

  //
  // Parse the command line to obtain GEMM dimensions and scalar values.
  //

  // GEMM problem dimensions.
  int problem[3] = { 128, 128, 131  };

  for (int i = 1; i < argc && i < 4; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  float scalars[2] = { 1, 0 };

  for (int i = 4; i < argc && i < 6; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 4];
  }

  //
  // Run the CUTLASS GEMM test.
  //

  hipError_t result = TestCutlassGemm(
    problem[0],     // GEMM M dimension
    problem[1],     // GEMM N dimension
    problem[2],     // GEMM K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
